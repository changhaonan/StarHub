#include "hip/hip_runtime.h"
#include <star/common/common_types.h>
#include <star/data_proc/depth_surfel_collect.h>
#include <>

namespace star
{
	namespace device
	{

		__global__ void markValidDepthPixelKernel(
			hipTextureObject_t vertex_img,
			const unsigned rows, const unsigned cols,
			PtrSz<char> valid_indicator)
		{
			const auto x = threadIdx.x + blockDim.x * blockIdx.x;
			const auto y = threadIdx.y + blockDim.y * blockIdx.y;
			if (x >= cols || y >= rows)
				return;

			// This indicator must finally written to output array
			const auto flatten_idx = x + cols * y;
			char valid = 0;
			const float4 vertex_confid = tex2D<float4>(vertex_img, x, y);
			if (vertex_confid.z > 0.f)
			{ // vertex_confid.z is equal to -depth, and this can cooperate with other filters
				valid = 1;
			}

			// Write to output
			valid_indicator[flatten_idx] = valid;
		}

		__global__ void collectDepthSurfelKernel(
			hipTextureObject_t vertex_confid_map,
			hipTextureObject_t normal_radius_map,
			hipTextureObject_t color_time_map,
			const PtrSz<const int> selected_array,
			const unsigned rows, const unsigned cols,
			PtrSz<DepthSurfel> valid_depth_surfel)
		{
			const auto selected_idx = threadIdx.x + blockDim.x * blockIdx.x;
			if (selected_idx >= selected_array.size)
				return;
			const auto idx = selected_array[selected_idx];
			const auto x = idx % cols;
			const auto y = idx / cols;

			// Construct the output
			DepthSurfel surfel;
			surfel.pixel_coord.x() = x;
			surfel.pixel_coord.y() = y;
			surfel.vertex_confid = tex2D<float4>(vertex_confid_map, x, y);
			surfel.normal_radius = tex2D<float4>(normal_radius_map, x, y);
			surfel.color_time = tex2D<float4>(color_time_map, x, y);

			// Write to the output array
			valid_depth_surfel[selected_idx] = surfel;
		}

	}; /* End of namespace deivce */
};	   /* End of namespace star */

void star::markValidDepthPixel(
	hipTextureObject_t vertex_img,
	const unsigned rows, const unsigned cols,
	GArray<char> &valid_indicator,
	hipStream_t stream)
{
	dim3 blk(16, 16);
	dim3 grid(divUp(cols, blk.x), divUp(rows, blk.y));
	device::markValidDepthPixelKernel<<<grid, blk, 0, stream>>>(
		vertex_img,
		rows, cols,
		valid_indicator);

	// Sync and check error
#if defined(CUDA_DEBUG_SYNC_CHECK)
	cudaSafeCall(hipStreamSynchronize(stream));
	cudaSafeCall(hipGetLastError());
#endif
}

void star::collectDepthSurfel(
	hipTextureObject_t vertex_confid_map,
	hipTextureObject_t normal_radius_map,
	hipTextureObject_t color_time_map,
	const GArray<int> &selected_array,
	const unsigned rows, const unsigned cols,
	GArray<DepthSurfel> &valid_depth_surfel,
	hipStream_t stream)
{
	dim3 blk(128);
	dim3 grid(divUp(selected_array.size(), blk.x));
	device::collectDepthSurfelKernel<<<grid, blk, 0, stream>>>(
		vertex_confid_map,
		normal_radius_map,
		color_time_map,
		selected_array,
		rows, cols,
		valid_depth_surfel);

	// Sync and check error
#if defined(CUDA_DEBUG_SYNC_CHECK)
	cudaSafeCall(hipStreamSynchronize(stream));
	cudaSafeCall(hipGetLastError());
#endif
}