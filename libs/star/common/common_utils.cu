#include <star/common/common_utils.h>
#include <star/common/safe_call_utils.hpp>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hip/driver_types.h>

hipCtx_t star::initCudaContext(int selected_device)
{
    // Initialize the cuda driver api
    cuSafeCall(hipInit(0));

    // Query the device
    int device_count = 0;
    cuSafeCall(hipGetDeviceCount(&device_count));
    for (auto dev_idx = 0; dev_idx < device_count; dev_idx++)
    {
        char dev_name[256] = {0};
        cuSafeCall(hipDeviceGetName(dev_name, 256, dev_idx));
        printf("device %d: %s\n", dev_idx, dev_name);
    }

    // Select the device
    printf("device %d is used as parallel processor.\n", selected_device);
    hipDevice_t cuda_device;
    cuSafeCall(hipDeviceGet(&cuda_device, selected_device));

    // Create cuda context
    hipCtx_t cuda_context;
    cuSafeCall(hipCtxCreate(&cuda_context, hipDeviceScheduleAuto, cuda_device));
    return cuda_context;
}

void star::destroyCudaContext(hipCtx_t context)
{
    hipDeviceSynchronize();
    cuSafeCall(hipCtxDestroy(context));
}
