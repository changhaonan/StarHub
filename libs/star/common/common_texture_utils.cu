#include <star/common/common_texture_utils.h>

hipTextureObject_t star::create1DLinearTexture(const GArray<float> &array)
{
	hipTextureDesc texture_desc;
	memset(&texture_desc, 0, sizeof(hipTextureDesc));
	texture_desc.normalizedCoords = 0;
	texture_desc.addressMode[0] = hipAddressModeBorder; // Return 0 outside the boundary
	texture_desc.addressMode[1] = hipAddressModeBorder;
	texture_desc.addressMode[2] = hipAddressModeBorder;
	texture_desc.filterMode = hipFilterModePoint;
	texture_desc.readMode = hipReadModeElementType;
	texture_desc.sRGB = 0;

	// Create resource desc
	hipResourceDesc resource_desc;
	memset(&resource_desc, 0, sizeof(hipResourceDesc));
	resource_desc.resType = hipResourceTypeLinear;
	resource_desc.res.linear.devPtr = (void *)array.ptr();
	resource_desc.res.linear.sizeInBytes = array.sizeBytes();
	resource_desc.res.linear.desc.f = hipChannelFormatKindFloat;
	resource_desc.res.linear.desc.x = 32;
	resource_desc.res.linear.desc.y = 0;
	resource_desc.res.linear.desc.z = 0;
	resource_desc.res.linear.desc.w = 0;

	// Allocate the texture
	hipTextureObject_t d_texture;
	cudaSafeCall(hipCreateTextureObject(&d_texture, &resource_desc, &texture_desc, nullptr));
	return d_texture;
}

hipTextureObject_t star::create1DLinearTexture(const GBufferArray<float> &array)
{
	GArray<float> pcl_array((float *)array.Ptr(), array.BufferSize());
	return create1DLinearTexture(pcl_array);
}

void star::createDefault2DTextureDesc(hipTextureDesc &desc)
{
	memset(&desc, 0, sizeof(desc));
	desc.addressMode[0] = hipAddressModeBorder; // Return 0 outside the boundary
	desc.addressMode[1] = hipAddressModeBorder;
	desc.addressMode[2] = hipAddressModeBorder;
	desc.filterMode = hipFilterModePoint;
	desc.readMode = hipReadModeElementType;
	desc.normalizedCoords = 0;
}

void star::createDefault3DTextureDesc(hipTextureDesc &desc)
{
	memset(&desc, 0, sizeof(desc));
	desc.addressMode[0] = hipAddressModeBorder; // Return 0 outside the boundary
	desc.addressMode[1] = hipAddressModeBorder;
	desc.addressMode[2] = hipAddressModeBorder;
	desc.filterMode = hipFilterModePoint;
	desc.readMode = hipReadModeElementType;
	desc.normalizedCoords = 0;
}

void star::createDepthTexture(
	const unsigned img_rows,
	const unsigned img_cols,
	hipTextureObject_t &texture,
	hipArray_t &d_array)
{
	// The texture description
	hipTextureDesc depth_texture_desc;
	createDefault2DTextureDesc(depth_texture_desc);

	// Create channel descriptions
	hipChannelFormatDesc depth_channel_desc = hipCreateChannelDesc(16, 0, 0, 0, hipChannelFormatKindUnsigned);

	// Allocate the cuda array
	cudaSafeCall(hipMallocArray(&d_array, &depth_channel_desc, img_cols, img_rows));

	// Create the resource desc
	hipResourceDesc resource_desc;
	memset(&resource_desc, 0, sizeof(hipResourceDesc));
	resource_desc.resType = hipResourceTypeArray;
	resource_desc.res.array.array = d_array;

	// Allocate the texture
	cudaSafeCall(hipCreateTextureObject(&texture, &resource_desc, &depth_texture_desc, 0));
}

void star::createDepthTextureSurface(
	const unsigned img_rows,
	const unsigned img_cols,
	hipTextureObject_t &texture,
	hipSurfaceObject_t &surface,
	hipArray_t &d_array)
{
	// The texture description
	hipTextureDesc depth_texture_desc;
	createDefault2DTextureDesc(depth_texture_desc);

	// Create channel descriptions
	hipChannelFormatDesc depth_channel_desc = hipCreateChannelDesc(16, 0, 0, 0, hipChannelFormatKindUnsigned);

	// Allocate the cuda array
	cudaSafeCall(hipMallocArray(&d_array, &depth_channel_desc, img_cols, img_rows));

	// Create the resource desc
	hipResourceDesc resource_desc;
	memset(&resource_desc, 0, sizeof(hipResourceDesc));
	resource_desc.resType = hipResourceTypeArray;
	resource_desc.res.array.array = d_array;

	// Allocate the texture
	cudaSafeCall(hipCreateTextureObject(&texture, &resource_desc, &depth_texture_desc, 0));
	cudaSafeCall(hipCreateSurfaceObject(&surface, &resource_desc));
}

void star::createDepthTextureSurface(const unsigned img_rows, const unsigned img_cols, CudaTextureSurface &collect)
{
	STAR_CHECK_NE(img_rows, 0);
	STAR_CHECK_NE(img_cols, 0);
	createDepthTextureSurface(
		img_rows, img_cols,
		collect.texture, collect.surface, collect.d_array);
}

void star::createIndexTextureSurface(
	const unsigned img_rows,
	const unsigned img_cols,
	hipTextureObject_t &texture,
	hipSurfaceObject_t &surface,
	hipArray_t &d_array)
{
	// The texture description
	hipTextureDesc index_texture_desc;
	createDefault2DTextureDesc(index_texture_desc);

	// Create channel descriptions
	hipChannelFormatDesc index_channel_desc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindUnsigned);

	// Allocate the cuda array
	cudaSafeCall(hipMallocArray(&d_array, &index_channel_desc, img_cols, img_rows));

	// Create the resource desc
	hipResourceDesc resource_desc;
	memset(&resource_desc, 0, sizeof(hipResourceDesc));
	resource_desc.resType = hipResourceTypeArray;
	resource_desc.res.array.array = d_array;

	// Allocate the texture
	cudaSafeCall(hipCreateTextureObject(&texture, &resource_desc, &index_texture_desc, 0));
	cudaSafeCall(hipCreateSurfaceObject(&surface, &resource_desc));
}

void star::createIndexTextureSurface(const unsigned img_rows, const unsigned img_cols, CudaTextureSurface &collect)
{
	STAR_CHECK_NE(img_rows, 0);
	STAR_CHECK_NE(img_cols, 0);
	createIndexTextureSurface(
		img_rows, img_cols,
		collect.texture, collect.surface, collect.d_array);
}

void star::createInt32TextureSurface(
	const unsigned img_rows, const unsigned img_cols,
	hipTextureObject_t &texture, hipSurfaceObject_t &surface,
	hipArray_t &d_array)
{
	// The texture description
	hipTextureDesc int32_texture_desc;
	createDefault2DTextureDesc(int32_texture_desc);

	// Create channel descriptions
	hipChannelFormatDesc int32_channel_desc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindSigned);

	// Allocate the cuda array
	cudaSafeCall(hipMallocArray(&d_array, &int32_channel_desc, img_cols, img_rows));

	// Create the resource desc
	hipResourceDesc resource_desc;
	memset(&resource_desc, 0, sizeof(hipResourceDesc));
	resource_desc.resType = hipResourceTypeArray;
	resource_desc.res.array.array = d_array;

	// Allocate the texture
	cudaSafeCall(hipCreateTextureObject(&texture, &resource_desc, &int32_texture_desc, 0));
	cudaSafeCall(hipCreateSurfaceObject(&surface, &resource_desc));
}

void star::createInt32TextureSurface(
	const unsigned img_rows, const unsigned img_cols,
	CudaTextureSurface &collect)
{
	STAR_CHECK_NE(img_rows, 0);
	STAR_CHECK_NE(img_cols, 0);
	createInt32TextureSurface(
		img_rows, img_cols,
		collect.texture, collect.surface, collect.d_array);
}

void star::createFloat4TextureSurface(
	const unsigned rows, const unsigned cols,
	hipTextureObject_t &texture,
	hipSurfaceObject_t &surface,
	hipArray_t &d_array)
{
	// The texture description
	hipTextureDesc float4_texture_desc;
	createDefault2DTextureDesc(float4_texture_desc);

	// Create channel descriptions
	hipChannelFormatDesc float4_channel_desc = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);

	// Allocate the cuda array
	cudaSafeCall(hipMallocArray(&d_array, &float4_channel_desc, cols, rows));

	// Create the resource desc
	hipResourceDesc resource_desc;
	memset(&resource_desc, 0, sizeof(hipResourceDesc));
	resource_desc.resType = hipResourceTypeArray;
	resource_desc.res.array.array = d_array;

	// Allocate the texture
	cudaSafeCall(hipCreateTextureObject(&texture, &resource_desc, &float4_texture_desc, 0));
	cudaSafeCall(hipCreateSurfaceObject(&surface, &resource_desc));
}

void star::createFloat4TextureSurface(const unsigned rows, const unsigned cols, CudaTextureSurface &texture_collect)
{
	STAR_CHECK_NE(rows, 0);
	STAR_CHECK_NE(cols, 0);
	createFloat4TextureSurface(
		rows, cols,
		texture_collect.texture,
		texture_collect.surface,
		texture_collect.d_array);
}

void star::createFloat3TextureSurface(
	const unsigned rows, const unsigned cols,
	hipTextureObject_t &texture,
	hipSurfaceObject_t &surface,
	hipArray_t &d_array)
{
	// The texture description
	hipTextureDesc float3_texture_desc;
	createDefault2DTextureDesc(float3_texture_desc);

	// Create channel descriptions
	hipChannelFormatDesc float3_channel_desc = hipCreateChannelDesc(32, 32, 32, 0, hipChannelFormatKindFloat);

	// Allocate the cuda array
	cudaSafeCall(hipMallocArray(&d_array, &float3_channel_desc, cols, rows));

	// Create the resource desc
	hipResourceDesc resource_desc;
	memset(&resource_desc, 0, sizeof(hipResourceDesc));
	resource_desc.resType = hipResourceTypeArray;
	resource_desc.res.array.array = d_array;

	// Allocate the texture
	cudaSafeCall(hipCreateTextureObject(&texture, &resource_desc, &float3_texture_desc, 0));
	cudaSafeCall(hipCreateSurfaceObject(&surface, &resource_desc));
}

void star::createFloat3TextureSurface(
	const unsigned rows, const unsigned cols,
	CudaTextureSurface &texture_collect)
{
	STAR_CHECK_NE(rows, 0);
	STAR_CHECK_NE(cols, 0);
	createFloat3TextureSurface(
		rows, cols,
		texture_collect.texture,
		texture_collect.surface,
		texture_collect.d_array);
}

void star::createFloat2TextureSurface(
	const unsigned rows, const unsigned cols,
	hipTextureObject_t &texture,
	hipSurfaceObject_t &surface,
	hipArray_t &d_array)
{
	// The texture description
	hipTextureDesc float2_texture_desc;
	createDefault2DTextureDesc(float2_texture_desc);

	// Create channel descriptions
	hipChannelFormatDesc float2_channel_desc = hipCreateChannelDesc(32, 32, 0, 0, hipChannelFormatKindFloat);

	// Allocate the cuda array
	cudaSafeCall(hipMallocArray(&d_array, &float2_channel_desc, cols, rows));

	// Create the resource desc
	hipResourceDesc resource_desc;
	memset(&resource_desc, 0, sizeof(hipResourceDesc));
	resource_desc.resType = hipResourceTypeArray;
	resource_desc.res.array.array = d_array;

	// Allocate the texture
	cudaSafeCall(hipCreateTextureObject(&texture, &resource_desc, &float2_texture_desc, 0));
	cudaSafeCall(hipCreateSurfaceObject(&surface, &resource_desc));
}

void star::createFloat2TextureSurface(
	const unsigned rows, const unsigned cols,
	CudaTextureSurface &texture_collect)
{
	STAR_CHECK_NE(rows, 0);
	STAR_CHECK_NE(cols, 0);
	createFloat2TextureSurface(
		rows, cols,
		texture_collect.texture,
		texture_collect.surface,
		texture_collect.d_array);
}

void star::createFloat1TextureSurface(
	const unsigned rows, const unsigned cols,
	hipTextureObject_t &texture,
	hipSurfaceObject_t &surface,
	hipArray_t &d_array,
	const bool interpolationable)
{
	// The texture description
	hipTextureDesc float1_texture_desc;
	createDefault2DTextureDesc(float1_texture_desc);

	if (interpolationable)
	{
		float1_texture_desc.filterMode = hipFilterModeLinear;
	}
	else
	{
		// Deafault point
	}

	// Create channel descriptions
	hipChannelFormatDesc float1_channel_desc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);

	// Allocate the cuda array
	cudaSafeCall(hipMallocArray(&d_array, &float1_channel_desc, cols, rows));

	// Create the resource desc
	hipResourceDesc resource_desc;
	memset(&resource_desc, 0, sizeof(hipResourceDesc));
	resource_desc.resType = hipResourceTypeArray;
	resource_desc.res.array.array = d_array;

	// Allocate the texture
	cudaSafeCall(hipCreateTextureObject(&texture, &resource_desc, &float1_texture_desc, 0));
	cudaSafeCall(hipCreateSurfaceObject(&surface, &resource_desc));
}

void star::createFloat1TextureSurface(
	const unsigned rows, const unsigned cols,
	CudaTextureSurface &texture_collect,
	const bool interpolationable)
{
	STAR_CHECK_NE(rows, 0);
	STAR_CHECK_NE(cols, 0);
	createFloat1TextureSurface(
		rows, cols,
		texture_collect.texture,
		texture_collect.surface,
		texture_collect.d_array,
		interpolationable);
}

void star::createUChar1TextureSurface(
	const unsigned rows, const unsigned cols,
	hipTextureObject_t &texture,
	hipSurfaceObject_t &surface,
	hipArray_t &d_array)
{
	// The texture description
	hipTextureDesc uchar1_texture_desc;
	createDefault2DTextureDesc(uchar1_texture_desc);

	// Create channel descriptions
	hipChannelFormatDesc uchar1_channel_desc = hipCreateChannelDesc(8, 0, 0, 0, hipChannelFormatKindUnsigned);

	// Allocate the cuda array
	cudaSafeCall(hipMallocArray(&d_array, &uchar1_channel_desc, cols, rows));

	// Create the resource desc
	hipResourceDesc resource_desc;
	memset(&resource_desc, 0, sizeof(hipResourceDesc));
	resource_desc.resType = hipResourceTypeArray;
	resource_desc.res.array.array = d_array;

	// Allocate the texture
	cudaSafeCall(hipCreateTextureObject(&texture, &resource_desc, &uchar1_texture_desc, 0));
	cudaSafeCall(hipCreateSurfaceObject(&surface, &resource_desc));
}

void star::createUChar1TextureSurface(
	const unsigned rows, const unsigned cols,
	CudaTextureSurface &texture_collect)
{
	STAR_CHECK_NE(rows, 0);
	STAR_CHECK_NE(cols, 0);
	createUChar1TextureSurface(
		rows, cols,
		texture_collect.texture,
		texture_collect.surface,
		texture_collect.d_array);
}

void star::query2DTextureExtent(hipTextureObject_t texture, unsigned &width, unsigned &height)
{
	hipResourceDesc texture_res;
	cudaSafeCall(hipGetTextureObjectResourceDesc(&texture_res, texture));
	hipArray_t cu_array = texture_res.res.array.array;
	hipChannelFormatDesc channel_desc;
	hipExtent extent;
	unsigned int flag;
	cudaSafeCall(hipArrayGetInfo(&channel_desc, &extent, &flag, cu_array));

	width = extent.width;
	height = extent.height;
}

void star::releaseTextureCollect(CudaTextureSurface &texture_collect)
{
	cudaSafeCall(hipDestroyTextureObject(texture_collect.texture));
	cudaSafeCall(hipDestroySurfaceObject(texture_collect.surface));
	cudaSafeCall(hipFreeArray(texture_collect.d_array));
}

void star::createFloat3DTextureSurface(
	const unsigned cols, const unsigned rows, const unsigned height,
	hipTextureObject_t &texture,
	hipSurfaceObject_t &surface,
	hipArray_t &d_array,
	const bool interpolationable)
{

	// The texture description
	hipTextureDesc float1_texture_desc;
	createDefault3DTextureDesc(float1_texture_desc);

	if (interpolationable)
	{
		float1_texture_desc.filterMode = hipFilterModeLinear;
	}
	else
	{
		// Deafault point
	}

	// Create channel descriptions
	hipChannelFormatDesc float1_channel_desc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);

	// Allocate the cuda array
	cudaSafeCall(hipMalloc3DArray(&d_array, &float1_channel_desc, make_hipExtent(cols, rows, height)));

	// Create the resource desc
	hipResourceDesc resource_desc;
	memset(&resource_desc, 0, sizeof(hipResourceDesc));
	resource_desc.resType = hipResourceTypeArray;
	resource_desc.res.array.array = d_array;

	// Allocate the texture
	cudaSafeCall(hipCreateTextureObject(&texture, &resource_desc, &float1_texture_desc, 0));
	cudaSafeCall(hipCreateSurfaceObject(&surface, &resource_desc));
}

void star::createFloat3DTextureSurface(
	const unsigned cols, const unsigned rows, const unsigned height,
	CudaTextureSurface &texture_collect,
	const bool interpolationable)
{
	STAR_CHECK_NE(rows, 0);
	STAR_CHECK_NE(cols, 0);
	STAR_CHECK_NE(height, 0);
	createFloat3DTextureSurface(
		cols, rows, height,
		texture_collect.texture,
		texture_collect.surface,
		texture_collect.d_array,
		interpolationable);
}

void star::createUchar3DTextureSurface(
	const unsigned cols, const unsigned rows, const unsigned height,
	hipTextureObject_t &texture,
	hipSurfaceObject_t &surface,
	hipArray_t &d_array)
{
	// The texture description
	hipTextureDesc uchar_texture_desc;
	createDefault3DTextureDesc(uchar_texture_desc);

	// Create channel descriptions
	hipChannelFormatDesc uchar_channel_desc = hipCreateChannelDesc(8, 0, 0, 0, hipChannelFormatKindUnsigned);

	// Allocate the cuda array
	cudaSafeCall(hipMalloc3DArray(&d_array, &uchar_channel_desc, make_hipExtent(cols, rows, height)));

	// Create the resource desc
	hipResourceDesc resource_desc;
	memset(&resource_desc, 0, sizeof(hipResourceDesc));
	resource_desc.resType = hipResourceTypeArray;
	resource_desc.res.array.array = d_array;

	// Allocate the texture
	cudaSafeCall(hipCreateTextureObject(&texture, &resource_desc, &uchar_texture_desc, 0));
	cudaSafeCall(hipCreateSurfaceObject(&surface, &resource_desc));
}

void star::createUchar3DTextureSurface(
	const unsigned cols, const unsigned rows, const unsigned height,
	CudaTextureSurface &texture_collect)
{
	STAR_CHECK_NE(rows, 0);
	STAR_CHECK_NE(cols, 0);
	STAR_CHECK_NE(height, 0);
	createUchar3DTextureSurface(
		cols, rows, height,
		texture_collect.texture,
		texture_collect.surface,
		texture_collect.d_array);
}

void star::query3DTextureExtent(
	hipTextureObject_t texture, unsigned &width, unsigned &height, unsigned &depth)
{
	hipResourceDesc texture_res;
	cudaSafeCall(hipGetTextureObjectResourceDesc(&texture_res, texture));
	hipArray_t cu_array = texture_res.res.array.array;
	hipChannelFormatDesc channel_desc;
	hipExtent extent;
	unsigned int flag;
	cudaSafeCall(hipArrayGetInfo(&channel_desc, &extent, &flag, cu_array));

	width = extent.width;
	height = extent.height;
	depth = extent.depth;
}