#include "KeyPoints.h"

star::KeyPoints::KeyPoints(const KeyPointType keypoint_type)
    : SurfelGeometrySC(), m_keypoint_type(keypoint_type)
{
    switch (keypoint_type)
    {
    case KeyPointType::R2D2:
        m_descriptor_dim = 128;
        break;
    case KeyPointType::SuperPoints:
        m_descriptor_dim = 256;
        break;
    default:
        break;
    }

    m_descriptor.AllocateBuffer(d_max_num_keypoints * m_descriptor_dim);
}

star::KeyPoints::~KeyPoints()
{
    m_descriptor.ReleaseBuffer();
}

void star::KeyPoints::Resize(size_t size)
{
    ResizeValidSurfelArrays(size);
    m_descriptor.ResizeArrayOrException(size * m_descriptor_dim);
}

void star::KeyPoints::ReAnchor(
    KeyPoints::ConstPtr src_keypoints,
    KeyPoints::Ptr tar_keypoints,
    hipStream_t stream)
{
    // Copy owned data
    cudaSafeCall(
        hipMemcpyAsync(
            tar_keypoints->SurfelKNN(),
            src_keypoints->SurfelKNNReadOnly(),
            src_keypoints->SurfelKNNReadOnly().ByteSize(),
            hipMemcpyDeviceToDevice, stream));
    cudaSafeCall(
        hipMemcpyAsync(
            tar_keypoints->SurfelKNNSpatialWeight(),
            src_keypoints->SurfelKNNSpatialWeightReadOnly(),
            src_keypoints->SurfelKNNSpatialWeightReadOnly().ByteSize(),
            hipMemcpyDeviceToDevice, stream));
    cudaSafeCall(
        hipMemcpyAsync(
            tar_keypoints->SurfelKNNConnectWeight(),
            src_keypoints->SurfelKNNConnectWeightReadOnly(),
            src_keypoints->SurfelKNNConnectWeightReadOnly().ByteSize(),
            hipMemcpyDeviceToDevice, stream));
    // Copy geometry data
    cudaSafeCall(
        hipMemcpyAsync(
            tar_keypoints->ReferenceVertexConfidence(),
            src_keypoints->LiveVertexConfidenceReadOnly(),
            src_keypoints->LiveVertexConfidenceReadOnly().ByteSize(),
            hipMemcpyDeviceToDevice, stream));
    cudaSafeCall(
        hipMemcpyAsync(
            tar_keypoints->ReferenceNormalRadius(),
            src_keypoints->LiveNormalRadiusReadOnly(),
            src_keypoints->LiveNormalRadiusReadOnly().ByteSize(),
            hipMemcpyDeviceToDevice, stream));
    cudaSafeCall(
        hipMemcpyAsync(
            tar_keypoints->ColorTime(),
            src_keypoints->ColorTimeReadOnly(),
            src_keypoints->ColorTimeReadOnly().ByteSize(),
            hipMemcpyDeviceToDevice, stream));
    // Optional
    cudaSafeCall(
        hipMemcpyAsync(
            tar_keypoints->SemanticProb(),
            src_keypoints->SemanticProbReadOnly(),
            src_keypoints->SemanticProbReadOnly().ByteSize(),
            hipMemcpyDeviceToDevice, stream));
    // Descriptor
    cudaSafeCall(
        hipMemcpyAsync(
            tar_keypoints->Descriptor(),
            src_keypoints->DescriptorReadOnly(),
            src_keypoints->DescriptorReadOnly().ByteSize(),
            hipMemcpyDeviceToDevice, stream));

    // Sync & Resize
    cudaSafeCall(hipStreamSynchronize(stream));
    tar_keypoints->Resize(src_keypoints->NumKeyPoints());
}