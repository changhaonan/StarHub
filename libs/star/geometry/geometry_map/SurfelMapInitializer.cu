#include "hip/hip_runtime.h"
#include <star/geometry/geometry_map/SurfelMapInitializer.h>
#include <star/img_proc/surfel_reliable_filter.h>
#include <star/img_proc/generate_maps.h>

namespace star::device
{
    constexpr float d_max_measure_clip = 10.f;

    __global__ void ExtractValidTsdfCenterKernel(
        hipTextureObject_t tsdf_val,
        float4 *__restrict__ valid_tsdf_center,
        unsigned *__restrict__ valid_count,
        const unsigned width,
        const unsigned height,
        const unsigned depth,
        const float voxel_size,
        const float3 origin,
        const unsigned max_num_surfels)
    {
        const int x = threadIdx.x + blockDim.x * blockIdx.x;
        const int y = threadIdx.y + blockDim.y * blockIdx.y;
        const int z = threadIdx.z + blockDim.z * blockIdx.z;
        if (x >= width || y >= height || z >= depth)
            return;
        float tsdf = tex3D<float>(tsdf_val, float(x) + 0.5f, float(y) + 0.5f, float(z) + 0.5f);
        if (fabs(tsdf) > 1e-6f && fabs(tsdf) < 1.f && (tsdf < 0.f))
        { // Valid part. Not counting 1
            unsigned offset = atomicAdd(valid_count, 1);
            if (offset < max_num_surfels)
            {
                valid_tsdf_center[offset] = make_float4(
                    float(x) * voxel_size + origin.x,
                    float(y) * voxel_size + origin.y,
                    float(z) * voxel_size + origin.z,
                    1.f);
            }
            else
            {
                printf("Offset is %d, exceed %d.\n", offset, max_num_surfels);
            }
        }
    }

    __global__ void ComputeScaledVertexFromDepthKernel(
        const hipTextureObject_t depth_img_collect, // In (m)
        const hipTextureObject_t raw_depth_img_collect,
        GArraySlice<float4> measurement_surfel, // In (m)
        const unsigned img_rows,
        const unsigned img_cols,
        const Intrinsic intrinsic)
    {
        const int x = threadIdx.x + blockDim.x * blockIdx.x;
        const int y = threadIdx.y + blockDim.y * blockIdx.y;
        if (x >= img_cols || y >= img_rows)
            return;

        unsigned p_idx = x + y * img_cols;
        float depth_measure = tex2D<float>(depth_img_collect, float(x) + 0.5f, float(y) + 0.5f);
        unsigned short raw_depth_measure = tex2D<unsigned short>(raw_depth_img_collect, float(x), float(y));
        float4 measure_p;
        if (fabs(depth_measure) < 1e-6f || fabs(depth_measure) > d_max_measure_clip)
        { // Upper-bound: 10 m
            measure_p = make_float4(0.f, 0.f, 0.f, 1.f);
        }
        else
        {
            measure_p.x = (float(x) - intrinsic.principal_x) / intrinsic.focal_x * depth_measure;
            measure_p.y = (float(y) - intrinsic.principal_y) / intrinsic.focal_y * depth_measure;
            measure_p.z = depth_measure; // z = 0 will be put to Z-surface
            measure_p.w = 1.f;
        }

        measurement_surfel[p_idx] = measure_p;
    }

    __global__ void ComputeScaledVertexFromDepthKernel(
        const hipTextureObject_t depth_img_collect,   // In (m)
        hipSurfaceObject_t measurement_vertex_confid, // In (m)
        const unsigned img_rows,
        const unsigned img_cols,
        const Intrinsic intrinsic,
        const float downsample_scale_inv)
    {
        const int x = threadIdx.x + blockDim.x * blockIdx.x;
        const int y = threadIdx.y + blockDim.y * blockIdx.y;
        if (x >= img_cols || y >= img_rows)
            return;

        float depth_measure = tex2D<float>(depth_img_collect, float(x) * downsample_scale_inv + 0.5f, float(y) * downsample_scale_inv + 0.5f);
        float4 measure_p;
        if (fabs(depth_measure) < 1e-6f || fabs(depth_measure) > d_max_measure_clip)
        { // Upper-bound: 10 m
            measure_p = make_float4(0.f, 0.f, 0.f, 1.f);
        }
        else
        {
            measure_p.x = (float(x) - intrinsic.principal_x) / intrinsic.focal_x * depth_measure;
            measure_p.y = (float(y) - intrinsic.principal_y) / intrinsic.focal_y * depth_measure;
            measure_p.z = depth_measure; // z = 0 will be put to Z-surface
            measure_p.w = 1.f;
        }

        // Write
        surf2Dwrite(measure_p, measurement_vertex_confid, x * sizeof(float4), y);
    }

    __global__ void filterAndScaleVertexKernel(
        hipTextureObject_t raw_depth_map,
        hipTextureObject_t raw_vertex_confid_map,
        hipSurfaceObject_t filtered_vertex_confid_map,
        const unsigned scaled_img_cols,
        const unsigned scaled_img_rows,
        const float downsample_scale_inv,
        const float clip_near,
        const float clip_far)
    {
        const int x = threadIdx.x + blockDim.x * blockIdx.x;
        const int y = threadIdx.y + blockDim.y * blockIdx.y;
        if (x >= scaled_img_cols || y >= scaled_img_rows)
            return;
        const int raw_x = x * downsample_scale_inv;
        const int raw_y = y * downsample_scale_inv;

        const auto depth = tex2D<float>(raw_depth_map, raw_x, raw_y); // depth should be in clamp mode
        float4 vertex_confid;
        if (abs(depth) > clip_far || abs(depth) < clip_near)
        { // Invalid pixel
            vertex_confid = make_float4(0.f, 0.f, 0.f, 0.f);
        }
        else
        {
            vertex_confid = tex2D<float4>(raw_vertex_confid_map, raw_x, raw_y);
        }

        // Write: allow for local replacing
        surf2Dwrite(vertex_confid, filtered_vertex_confid_map, x * sizeof(float4), y);
    }

    // Valid is decideded by the vertex confid
    __global__ void ComputeIndexMapKernel(
        hipTextureObject_t vertex_confid_map,
        hipSurfaceObject_t index_map,
        unsigned *valid_count,
        const unsigned index_offset,
        const unsigned img_cols,
        const unsigned img_rows)
    {
        const int x = threadIdx.x + blockDim.x * blockIdx.x;
        const int y = threadIdx.y + blockDim.y * blockIdx.y;
        if (x >= img_cols || y >= img_rows)
            return;

        float4 vertex_confid = tex2D<float4>(vertex_confid_map, x, y);
        if (fabs(vertex_confid.z) > 1e-8f)
        {
            unsigned old_count = atomicAdd(valid_count, (unsigned)1);
            surf2Dwrite(old_count + index_offset, index_map, x * sizeof(unsigned), y);
        }
        else
        {
            surf2Dwrite(0xffffffff, index_map, x * sizeof(unsigned), y); // unsigned max
        }
    }
}

star::SurfelMapInitializer::SurfelMapInitializer(
    const unsigned width, const unsigned height,
    const float clip_near, const float clip_far,
    const float surfel_radius_scale, const Intrinsic &intrinsic) : m_width(width), m_height(height),
                                                                   m_clip_near(clip_near), m_clip_far(clip_far),
                                                                   m_surfel_radius_scale(surfel_radius_scale),
                                                                   m_intrinsic(intrinsic)
{
    createDepthTextureSurface(height, width, m_raw_depth_img_collect);
    createFloat1TextureSurface(height, width, m_filtered_depth_img_collect);
    createFloat4TextureSurface(height, width, m_raw_vertex_confid);
    cudaSafeCall(hipMalloc((void **)&m_valid_count, sizeof(unsigned)));
}

star::SurfelMapInitializer::~SurfelMapInitializer()
{
    releaseTextureCollect(m_raw_depth_img_collect);
    releaseTextureCollect(m_filtered_depth_img_collect);
    releaseTextureCollect(m_raw_vertex_confid);
    cudaSafeCall(hipFree(m_valid_count));
}

void star::SurfelMapInitializer::UploadDepthImage(
    const GArrayView<unsigned short> depth_image,
    hipStream_t stream)
{
    // 1. Upload depth (Raw size)
    cudaSafeCall(hipMemcpy2DToArrayAsync(
        m_raw_depth_img_collect.d_array,
        0, 0,
        depth_image.Ptr(),
        m_width * sizeof(unsigned short),
        m_width * sizeof(unsigned short),
        m_height,
        hipMemcpyDeviceToDevice,
        stream));
    // 2. Filter depth
    filterUnreliableDepth(
        m_raw_depth_img_collect.texture,
        m_filtered_depth_img_collect.surface,
        m_height, m_width,
        m_clip_near, m_clip_far,
        m_intrinsic,
        stream);
}

void star::SurfelMapInitializer::InitFromRGBDImage(
    const GArrayView<uchar3> color_image,
    const GArrayView<unsigned short> depth_image,
    const float init_time,
    SurfelMap &surfel_map,
    hipStream_t stream)
{
    // 0. Check the scale
    STAR_CHECK_EQ(color_image.Size(), m_width * m_height);
    STAR_CHECK_EQ(depth_image.Size(), m_width * m_height);
    const float scale = float(surfel_map.Width()) / float(m_width);
    STAR_CHECK_EQ(scale, float(surfel_map.Height()) / float(m_height));

    // 1. Upload Depth image
    UploadDepthImage(depth_image, stream);

    // 2. Create vertex from depth
    InitFromVertexNormalDepth(surfel_map, scale, stream);

    // 3. Compute color time
    createScaledColorTimeMap(
        color_image,
        m_height,
        m_width,
        scale,
        init_time,
        surfel_map.m_color_time.surface,
        stream);

    // 4. Compute rgbd image (RGBD map is filtered out, used for optical-flow)
    createScaledRGBDMap( // RGBD image, D is the inverse of depth
        color_image,
        m_filtered_depth_img_collect.texture,
        m_height,
        m_width,
        scale,
        m_clip_near, m_clip_far,
        surfel_map.m_rgbd.surface,
        stream);

    // 5. Compute index map
    unsigned num_valid_surfel = 0;
    computeIndexMap(
        surfel_map.m_vertex_confid.texture,
        surfel_map.m_index.surface,
        num_valid_surfel,
        0,
        stream);

    // 6. Create the depth map 
    // FIXME: The depth map is different from the one used in rgbd map
    createScaledDepthMap(
		m_filtered_depth_img_collect.texture,
		m_height,
        m_width,
		scale,
		surfel_map.m_depth.surface,
		stream);

    cudaSafeCall(hipStreamSynchronize(stream));
    surfel_map.m_num_valid_surfel = num_valid_surfel;
}

void star::SurfelMapInitializer::InitFromVertexNormalDepth(
    SurfelMap &surfel_map,
    const float scale,
    hipStream_t stream)
{
    // 1. Create vertex on raw scale
    computeRawVertexFromDepth(
        m_raw_vertex_confid.surface,
        m_intrinsic,
        stream);

    // 2. Filter out invalid pixel & rescale
    // For this operation, we allow for in-place texture-surface operation
    filterAndScaleVertex(
        m_filtered_depth_img_collect.texture,
        m_raw_vertex_confid.texture,
        surfel_map.m_vertex_confid.surface,
        scale,
        m_clip_near,
        m_clip_far,
        stream);

    // 3. Compute normal
    createNormalRadiusMap(
        surfel_map.m_vertex_confid.texture,
        surfel_map.Height(),
        surfel_map.Width(),
        surfel_map.m_normal_radius.surface,
        m_surfel_radius_scale,
        stream);
}

void star::SurfelMapInitializer::computeRawVertexFromDepth(
    hipSurfaceObject_t vertex_confid_buffer,
    const Intrinsic &intrinsic,
    hipStream_t stream)
{
    dim3 blk(16, 16);
    dim3 grid(divUp(m_width, blk.x), divUp(m_height, blk.y));
    device::ComputeScaledVertexFromDepthKernel<<<grid, blk, 0, stream>>>(
        m_filtered_depth_img_collect.texture, // In (m)
        vertex_confid_buffer,
        m_height,
        m_width,
        intrinsic,
        1.f); // No re-scale here
}

void star::SurfelMapInitializer::filterAndScaleVertex(
    hipTextureObject_t raw_depth_map,
    hipTextureObject_t raw_vertex_confid_map,
    hipSurfaceObject_t filtered_vertex_confid_map,
    const float scale,
    const float clip_near,
    const float clip_far,
    hipStream_t stream)
{
    float scale_inv = 1.f / scale;
    unsigned scaled_height = m_height * scale;
    unsigned scaled_width = m_width * scale;

    dim3 blk(16, 16);
    dim3 grid(divUp(scaled_width, blk.x), divUp(scaled_height, blk.y));
    device::filterAndScaleVertexKernel<<<grid, blk, 0, stream>>>(
        raw_depth_map,
        raw_vertex_confid_map,
        filtered_vertex_confid_map,
        scaled_width,
        scaled_height,
        scale_inv,
        clip_near,
        clip_far);

    // Sync and check error
#if defined(CUDA_DEBUG_SYNC_CHECK)
    cudaSafeCall(hipStreamSynchronize(stream));
    cudaSafeCall(hipGetLastError());
#endif
}

void star::SurfelMapInitializer::computeIndexMap(
    hipTextureObject_t vertex_confid_map,
    hipSurfaceObject_t index_map,
    unsigned &valid_surfel_num,
    const unsigned index_offset,
    hipStream_t stream)
{
    cudaSafeCall(hipMemsetAsync(m_valid_count, 0, sizeof(unsigned), stream));

    unsigned img_width;
    unsigned img_height;
    query2DTextureExtent(vertex_confid_map, img_width, img_height);

    dim3 blk(16, 16);
    dim3 grid(divUp(img_width, blk.x), divUp(img_height, blk.y));
    device::ComputeIndexMapKernel<<<grid, blk, 0, stream>>>(
        vertex_confid_map,
        index_map,
        m_valid_count,
        index_offset,
        img_width,
        img_height);

    cudaSafeCall(hipMemcpyAsync(&valid_surfel_num, m_valid_count, sizeof(unsigned), hipMemcpyDeviceToHost, stream));
}