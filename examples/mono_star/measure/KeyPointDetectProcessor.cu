#include "hip/hip_runtime.h"
#include "KeyPointDetectProcessor.h"
#include <star/geometry/surfel/SurfelGeometryInitializer.h>
#include <star/geometry/keypoint/KeyPointMatcher.h>
#include <star/visualization/Visualizer.h>

namespace star::device
{
    __global__ void GetMatchedPointKernel(
        const float4 *__restrict__ vertex_confid_src,
        const float4 *__restrict__ vertex_confid_dst,
        float4 *__restrict__ matched_vertex_confid_src,
        float4 *__restrict__ matched_vertex_confid_dst,
        int2 *__restrict__ matches,
        const unsigned num_matches)
    {
        const unsigned idx = threadIdx.x + blockIdx.x * blockDim.x;
        if (idx >= num_matches)
            return;

        const int2 match = matches[idx];
        matched_vertex_confid_src[idx] = vertex_confid_src[match.x];
        matched_vertex_confid_dst[idx] = vertex_confid_dst[match.y];
    }

    // Here rgbd is (-1, 1) transfer them to (0, 255)
    __global__ void RGBDToRGBKernel(
        hipTextureObject_t rgbd_tex,
        uchar3 *__restrict__ rgb,
        const unsigned img_width,
        const unsigned img_height)
    {
        const unsigned idx = threadIdx.x + blockIdx.x * blockDim.x;
        const unsigned idy = threadIdx.y + blockIdx.y * blockDim.y;
        if (idx >= img_width || idy >= img_height)
            return;
        float4 rbgd = tex2D<float4>(rgbd_tex, idx, idy);
        uchar3 rgb_uchar3;
        rgb_uchar3.x = (unsigned char)((rbgd.x + 1) * 127.5);
        rgb_uchar3.y = (unsigned char)((rbgd.y + 1) * 127.5);
        rgb_uchar3.z = (unsigned char)((rbgd.z + 1) * 127.5);
        rgb[idy * img_width + idx] = rgb_uchar3;
    }

}

star::KeyPointDetectProcessor::KeyPointDetectProcessor()
    : m_buffer_idx(0), m_num_valid_matches(0)
{
    std::cout << "Initilize KeyPointDetectProcessor..." << std::endl;
    auto &config = ConfigParser::Instance();
    m_keypoint_type = config.keypoint_type();

    m_fetcher = std::make_shared<VolumeDeformFileFetch>(config.data_path());

    // Camera parameters
    m_step_frame = config.step_frame();
    m_start_frame_idx = config.start_frame_idx();
    m_cam2world = config.extrinsic()[0];
    m_enable_semantic_surfel = config.enable_semantic_surfel();
    m_downsample_scale = config.downsample_scale();
    m_image_width = config.downsample_img_cols();
    m_image_height = config.downsample_img_rows();

    // Create host buffer
    cudaSafeCall(hipHostMalloc(&m_keypoint_buffer, sizeof(float2) * d_max_num_keypoints));
    cudaSafeCall(hipHostMalloc(&m_descriptor_buffer, sizeof(float) * d_max_num_keypoints * KeyPoints::GetDescriptorDim(m_keypoint_type)));

    unsigned num_pixels = m_image_width * m_image_height;
    cudaSafeCall(hipHostMalloc(&m_h_rgb, sizeof(uchar3) * num_pixels));
    m_g_rgb.create(num_pixels * sizeof(uchar3));

    m_g_keypoints.AllocateBuffer(d_max_num_keypoints);
    m_keypoint_matches.AllocateBuffer(d_max_num_keypoints);

    m_measure_keypoints = std::make_shared<star::KeyPoints>(config.keypoint_type());
    m_model_keypoints = std::make_shared<star::KeyPoints>(config.keypoint_type());

    m_matched_vertex_src.AllocateBuffer(d_max_num_keypoints);
    m_matched_vertex_dst.AllocateBuffer(d_max_num_keypoints);

    // Matching related
    m_kp_match_ratio_thresh = config.kp_match_ratio_thresh();
    m_kp_match_dist_thresh = config.kp_match_dist_thresh();

    // Vis
    m_enable_vis = config.enable_vis();
    m_pcd_size = config.pcd_size();
}

star::KeyPointDetectProcessor::~KeyPointDetectProcessor()
{
    cudaSafeCall(hipHostFree(m_keypoint_buffer));
    cudaSafeCall(hipHostFree(m_descriptor_buffer));
    cudaSafeCall(hipHostFree(m_h_rgb));
    m_g_rgb.release();

    m_g_keypoints.ReleaseBuffer();
    m_keypoint_matches.ReleaseBuffer();

    m_matched_vertex_src.ReleaseBuffer();
    m_matched_vertex_dst.ReleaseBuffer();
}

void star::KeyPointDetectProcessor::ProcessFrame(
    const SurfelMapTex &measure_surfel_map,
    const SurfelMapTex &model_surfel_map,
    const unsigned frame_idx,
    hipStream_t stream)
{
    // Detect Feature
    detectFeature(measure_surfel_map, m_keypoint_tar, m_descriptor_tar, stream);
    buildKeyPoints(measure_surfel_map, m_keypoint_tar, m_descriptor_tar, m_measure_keypoints, stream);
    cudaSafeCall(hipStreamSynchronize(stream));
    if (frame_idx > 0)
    {
        detectFeature(model_surfel_map, m_keypoint_src, m_descriptor_src, stream);
        buildKeyPoints(model_surfel_map, m_keypoint_src, m_descriptor_src, m_model_keypoints, stream);
        cudaSafeCall(hipStreamSynchronize(stream));
        // Match Feature
        float kp_match_pixel_dist = 20.f;
        MatchKeyPointsBFOpenCVHostOnly(
            m_keypoint_src,   // Model
            m_keypoint_tar,   // Measure
            m_descriptor_src, // Model
            m_descriptor_tar, // Measure
            m_keypoint_matches.Slice(),
            m_num_valid_matches,
            m_kp_match_ratio_thresh,
            kp_match_pixel_dist,
            stream);
        m_keypoint_matches.ResizeArrayOrException(m_num_valid_matches);
    }

    // Sync stream
    cudaSafeCall(hipStreamSynchronize(stream));

    // Save context
    if (m_enable_vis)
        saveContext(frame_idx, stream);
}

void star::KeyPointDetectProcessor::saveContext(
    unsigned frame_idx, hipStream_t stream)
{
    auto &context = easy3d::Context::Instance();
    context.addPointCloud("measure_keypoints", "", Eigen::Matrix4f::Identity(), m_pcd_size);
    visualize::SaveSemanticPointCloud(
        m_measure_keypoints->LiveVertexConfidenceReadOnly(),
        m_measure_keypoints->SemanticProbReadOnly(),
        visualize::default_semantic_color_dict,
        context.at("measure_keypoints"));

    if (m_model_keypoints->NumKeyPoints() > 0)
    {
        context.addPointCloud("model_keypoints", "", Eigen::Matrix4f::Identity(), m_pcd_size);
        visualize::SaveSemanticPointCloud(
            m_measure_keypoints->LiveVertexConfidenceReadOnly(),
            m_measure_keypoints->SemanticProbReadOnly(),
            visualize::default_semantic_color_dict,
            context.at("model_keypoints"));
    }

    // Draw the matched keypoints
    if (m_num_valid_matches > 0)
    {
        getMatchedKeyPoints(*m_model_keypoints, *m_measure_keypoints, stream);
        context.addPointCloud("matched_keypoints", "", Eigen::Matrix4f::Identity(), m_pcd_size);
        visualize::SaveMatchedPointCloud(
            m_matched_vertex_src.View(),
            m_matched_vertex_dst.View(),
            context.at("matched_keypoints"));
    }
}

void star::KeyPointDetectProcessor::getMatchedKeyPoints(
    const KeyPoints &keypoints_src,
    const KeyPoints &keypoints_dst,
    hipStream_t stream)
{
    // Resize
    m_matched_vertex_src.ResizeArrayOrException(m_num_valid_matches);
    m_matched_vertex_dst.ResizeArrayOrException(m_num_valid_matches);

    // Get matched keypoints
    dim3 blk(128);
    dim3 grid(divUp(m_num_valid_matches, blk.x));
    device::GetMatchedPointKernel<<<grid, blk, 0, stream>>>(
        keypoints_src.ReferenceVertexConfidenceReadOnly().Ptr(),
        keypoints_dst.ReferenceVertexConfidenceReadOnly().Ptr(),
        m_matched_vertex_src.Ptr(),
        m_matched_vertex_dst.Ptr(),
        m_keypoint_matches.Ptr(),
        m_num_valid_matches);
}

void star::KeyPointDetectProcessor::detectFeature(
    const SurfelMapTex &surfel_map, cv::Mat &keypoint, cv::Mat &descriptor, hipStream_t stream)
{
    // Detect feature
    if (m_keypoint_type == KeyPointType::ORB)
    {
        detectORBFeature(surfel_map.rgbd, keypoint, descriptor, stream);
    }
    else
    {
        throw std::runtime_error("KeyPoint Type not supported!");
    }
}

void star::KeyPointDetectProcessor::detectORBFeature(
    hipTextureObject_t rgbd_tex, cv::Mat &keypoint, cv::Mat &descriptor, hipStream_t stream)
{
    // Download the image from Texture
    dim3 blk(32, 32);
    dim3 grid(divUp(m_image_width, blk.x), divUp(m_image_height, blk.y));
    device::RGBDToRGBKernel<<<grid, blk, 0, stream>>>(
        rgbd_tex, m_g_rgb.ptr(), m_image_width, m_image_height);

    // Copy from device to host
    cudaSafeCall(hipMemcpyAsync(
        m_h_rgb, m_g_rgb.ptr(),
        sizeof(uchar3) * m_image_width * m_image_height,
        hipMemcpyDeviceToHost, stream));
    cudaSafeCall(hipStreamSynchronize(stream));

    cv::Mat image(m_image_height, m_image_width, CV_8UC3, m_h_rgb);
    // Convert to gray
    cv::Mat gray_image;
    cv::cvtColor(image, gray_image, cv::COLOR_RGB2GRAY);
    // Detect ORB features
    std::vector<cv::KeyPoint> keypoints;
    cv::Ptr<cv::Feature2D> orb = cv::ORB::create();
    orb->detectAndCompute(gray_image, cv::Mat(), keypoints, descriptor);
    // Save keypoints into mat
    keypoint = cv::Mat(keypoints.size(), 1, CV_32FC2);
    for (int i = 0; i < keypoints.size(); i++)
    {
        keypoint.at<float2>(i, 0) = make_float2(keypoints[i].pt.x, keypoints[i].pt.y);
    }

    // Log info
    LOG(INFO) << "Number of keypoints detected: " << keypoints.size();
}

void star::KeyPointDetectProcessor::buildKeyPoints(
    const SurfelMapTex &surfel_map,
    const cv::Mat &keypoint,
    const cv::Mat &descriptor,
    KeyPoints::Ptr keypoints,
    hipStream_t stream)
{
    // Load the keypoint and descriptor into gpu
    unsigned num_keypoints_detected = keypoint.rows;
    // CPU copy
    memcpy(m_keypoint_buffer, keypoint.data,
           sizeof(float2) * keypoint.total());
    memcpy(m_descriptor_buffer, descriptor.data,
           sizeof(unsigned char) * descriptor.total());

    // Copy to GPU
    cudaSafeCall(hipMemcpyAsync(
        m_g_keypoints.Ptr(),
        m_keypoint_buffer,
        keypoint.total() * sizeof(float2),
        hipMemcpyHostToDevice,
        stream));
    cudaSafeCall(hipMemcpyAsync(
        keypoints->Descriptor().Ptr(),
        m_descriptor_buffer,
        descriptor.total() * sizeof(unsigned char),
        hipMemcpyHostToDevice,
        stream));
    cudaSafeCall(hipStreamSynchronize(stream));
    // Resize
    m_g_keypoints.ResizeArrayOrException(num_keypoints_detected);
    keypoints->Resize(num_keypoints_detected);

    // Build 3d keypoint
    // Init keypoint geometry
    SurfelGeometryInitializer::InitFromGeometryMap(
        *keypoints,
        surfel_map,
        m_g_keypoints.View(),
        m_cam2world,
        m_enable_semantic_surfel,
        stream);
    cudaSafeCall(hipStreamSynchronize(stream));
}