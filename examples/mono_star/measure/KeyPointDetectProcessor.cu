#include "hip/hip_runtime.h"
#include "KeyPointDetectProcessor.h"
#include <star/geometry/surfel/SurfelGeometryInitializer.h>
#include <star/geometry/keypoint/KeyPointMatcher.h>
#include <star/visualization/Visualizer.h>

namespace star::device
{
    __global__ void GetMatchedPointKernel(
        const float4 *__restrict__ vertex_confid_src,
        const float4 *__restrict__ vertex_confid_dst,
        float4 *__restrict__ matched_vertex_confid_src,
        float4 *__restrict__ matched_vertex_confid_dst,
        int2 *__restrict__ matches,
        const unsigned num_matches)
    {
        const unsigned idx = threadIdx.x + blockIdx.x * blockDim.x;
        if (idx >= num_matches)
            return;

        const int2 match = matches[idx];
        matched_vertex_confid_src[idx] = vertex_confid_src[match.x];
        matched_vertex_confid_dst[idx] = vertex_confid_dst[match.y];
    }

    // Here rgbd is (-1, 1) transfer them to (0, 255)
    __global__ void RGBDToRGBKernel(
        hipTextureObject_t rgbd_tex,
        uchar3 *__restrict__ rgb,
        const unsigned img_width,
        const unsigned img_height)
    {
        const unsigned idx = threadIdx.x + blockIdx.x * blockDim.x;
        const unsigned idy = threadIdx.y + blockIdx.y * blockDim.y;
        if (idx >= img_width || idy >= img_height)
            return;
        float4 rbgd = tex2D<float4>(rgbd_tex, idx, idy);
        uchar3 rgb_uchar3;
        rgb_uchar3.x = (unsigned char)((rbgd.x + 1) * 127.5);
        rgb_uchar3.y = (unsigned char)((rbgd.y + 1) * 127.5);
        rgb_uchar3.z = (unsigned char)((rbgd.z + 1) * 127.5);
        rgb[idy * img_width + idx] = rgb_uchar3;
    }

}

star::KeyPointDetectProcessor::KeyPointDetectProcessor()
    : m_buffer_idx(0), m_num_valid_matches(0)
{
    std::cout << "Initilize KeyPointDetectProcessor..." << std::endl;
    auto &config = ConfigParser::Instance();
    m_keypoint_type = config.keypoint_type();

    m_fetcher = std::make_shared<VolumeDeformFileFetch>(config.data_path());

    // Camera parameters
    m_step_frame = config.step_frame();
    m_start_frame_idx = config.start_frame_idx();
    m_cam2world = config.extrinsic()[0];
    m_enable_semantic_surfel = config.enable_semantic_surfel();
    m_downsample_scale = config.downsample_scale();
    m_image_width = config.downsample_img_cols();
    m_image_height = config.downsample_img_rows();

    // Create host buffer
    cudaSafeCall(hipHostMalloc(&m_keypoint_buffer, sizeof(float) * 2 * d_max_num_keypoints));
    cudaSafeCall(hipHostMalloc(&m_descriptor_buffer, sizeof(float) * d_max_num_keypoints * KeyPoints::GetDescriptorDim(m_keypoint_type)));

    unsigned num_pixels = m_image_width * m_image_height;
    cudaSafeCall(hipHostMalloc(&m_h_rgb, sizeof(uchar3) * num_pixels));
    m_g_rgb.create(num_pixels * sizeof(uchar3));

    m_g_keypoints.AllocateBuffer(d_max_num_keypoints);
    m_keypoint_matches.AllocateBuffer(d_max_num_keypoints);

    m_detected_keypoints = std::make_shared<star::KeyPoints>(config.keypoint_type());

    m_matched_vertex_src.AllocateBuffer(d_max_num_keypoints);
    m_matched_vertex_dst.AllocateBuffer(d_max_num_keypoints);

    // Matching related
    m_kp_match_ratio_thresh = config.kp_match_ratio_thresh();
    m_kp_match_dist_thresh = config.kp_match_dist_thresh();

    // Vis
    m_enable_vis = config.enable_vis();
    m_pcd_size = config.pcd_size();
}

star::KeyPointDetectProcessor::~KeyPointDetectProcessor()
{
    cudaSafeCall(hipHostFree(m_keypoint_buffer));
    cudaSafeCall(hipHostFree(m_descriptor_buffer));
    cudaSafeCall(hipHostFree(m_h_rgb));
    m_g_rgb.release();

    m_g_keypoints.ReleaseBuffer();
    m_keypoint_matches.ReleaseBuffer();

    m_matched_vertex_src.ReleaseBuffer();
    m_matched_vertex_dst.ReleaseBuffer();
}

void star::KeyPointDetectProcessor::ProcessFrame(
    const SurfelMapTex &measure_surfel_map,
    const SurfelMapTex &model_surfel_map,
    const KeyPoints &model_keypoints,
    const unsigned frame_idx,
    hipStream_t stream)
{
    // Detect Feature
    detectFeature(measure_surfel_map, m_keypoint_tar, m_descriptor_tar, stream);
    if (frame_idx > 0)
    {
        detectFeature(model_surfel_map, m_keypoint_src, m_descriptor_src, stream);
        MatchKeyPointsBFOpenCVHostOnly(
            m_keypoint_src,   // Model
            m_keypoint_tar,   // Measure
            m_descriptor_src, // Model
            m_descriptor_tar, // Measure
            m_keypoint_matches.Slice(),
            m_num_valid_matches,
            m_kp_match_ratio_thresh,
            m_kp_match_dist_thresh,
            stream);
        m_keypoint_matches.ResizeArrayOrException(m_num_valid_matches);
    }

    const auto image_idx = size_t(frame_idx) * m_step_frame + m_start_frame_idx;
    m_fetcher->FetchKeypoint(0, image_idx, m_keypoint_tar, m_descriptor_tar, m_keypoint_type);

    // Scale keypoints according to downsample ratio
    m_keypoint_tar = m_keypoint_tar * m_downsample_scale;

    // Load the keypoint and descriptor into gpu
    unsigned num_keypoints_detected = m_keypoint_tar.rows;
    // CPU copy
    memcpy(m_keypoint_buffer, m_keypoint_tar.data,
           sizeof(float) * m_keypoint_tar.total());
    memcpy(m_descriptor_buffer, m_descriptor_tar.data,
           sizeof(float) * m_descriptor_tar.total());

    // Copy to GPU
    cudaSafeCall(hipMemcpyAsync(
        m_g_keypoints.Ptr(),
        m_keypoint_buffer,
        m_keypoint_tar.total() * sizeof(float),
        hipMemcpyHostToDevice,
        stream));
    cudaSafeCall(hipMemcpyAsync(
        m_detected_keypoints->Descriptor().Ptr(),
        m_descriptor_buffer,
        m_descriptor_tar.total() * sizeof(float),
        hipMemcpyHostToDevice,
        stream));

    // Resize
    m_g_keypoints.ResizeArrayOrException(num_keypoints_detected);
    m_detected_keypoints->Resize(num_keypoints_detected);

    // Build 3d keypoint
    // Init keypoint geometry
    SurfelGeometryInitializer::InitFromGeometryMap(
        *m_detected_keypoints,
        measure_surfel_map,
        m_g_keypoints.View(),
        m_cam2world,
        m_enable_semantic_surfel,
        stream);

    if (frame_idx != 0 && m_detected_keypoints->NumKeyPoints() > 0 && model_keypoints.NumKeyPoints() > 0)
    {
        // Apply matching between model keypoints and detected keypoints
        MatchKeyPointsBFOpenCV(
            *m_detected_keypoints,
            model_keypoints,
            m_keypoint_matches.Slice(),
            m_num_valid_matches,
            m_kp_match_ratio_thresh,
            m_kp_match_dist_thresh,
            stream);
        m_keypoint_matches.ResizeArrayOrException(m_num_valid_matches);
    }

    // Sync stream
    cudaSafeCall(hipStreamSynchronize(stream));

    // Save context
    if (m_enable_vis)
        saveContext(model_keypoints, frame_idx, stream);
}

void star::KeyPointDetectProcessor::saveContext(
    const KeyPoints &model_keypoints, unsigned frame_idx, hipStream_t stream)
{
    auto &context = easy3d::Context::Instance();
    context.addPointCloud("d_keypoints", "", Eigen::Matrix4f::Identity(), m_pcd_size);
    visualize::SavePointCloud(
        m_detected_keypoints->LiveVertexConfidenceReadOnly(),
        context.at("d_keypoints"));

    if (model_keypoints.NumKeyPoints() > 0)
    {
        context.addPointCloud("model_keypoints", "", Eigen::Matrix4f::Identity(), m_pcd_size);
        visualize::SavePointCloud(
            model_keypoints.LiveVertexConfidenceReadOnly(),
            context.at("model_keypoints"));
    }

    // Draw the matched keypoints
    if (m_num_valid_matches > 0)
    {
        getMatchedKeyPoints(*m_detected_keypoints, model_keypoints, stream);
        context.addPointCloud("matched_keypoints", "", Eigen::Matrix4f::Identity(), m_pcd_size);
        visualize::SaveMatchedPointCloud(
            m_matched_vertex_src.View(),
            m_matched_vertex_dst.View(),
            context.at("matched_keypoints"));
    }
}

void star::KeyPointDetectProcessor::getMatchedKeyPoints(
    const KeyPoints &keypoints_src,
    const KeyPoints &keypoints_dst,
    hipStream_t stream)
{
    // Resize
    m_matched_vertex_src.ResizeArrayOrException(m_num_valid_matches);
    m_matched_vertex_dst.ResizeArrayOrException(m_num_valid_matches);

    // Get matched keypoints
    dim3 blk(128);
    dim3 grid(divUp(m_num_valid_matches, blk.x));
    device::GetMatchedPointKernel<<<grid, blk, 0, stream>>>(
        keypoints_src.ReferenceVertexConfidenceReadOnly().Ptr(),
        keypoints_dst.ReferenceVertexConfidenceReadOnly().Ptr(),
        m_matched_vertex_src.Ptr(),
        m_matched_vertex_dst.Ptr(),
        m_keypoint_matches.Ptr(),
        m_num_valid_matches);
}

void star::KeyPointDetectProcessor::detectFeature(
    const SurfelMapTex &surfel_map, cv::Mat &keypoint, cv::Mat &descriptor, hipStream_t stream)
{
    if (m_keypoint_type == KeyPointType::ORB)
    {
        detectORBFeature(surfel_map.rgbd, keypoint, descriptor, stream);
    }
    else
    {
        throw std::runtime_error("KeyPoint Type not supported!");
    }
}

void star::KeyPointDetectProcessor::detectORBFeature(
    hipTextureObject_t rgbd_tex, cv::Mat &keypoint, cv::Mat &descriptor, hipStream_t stream)
{
    // Download the image from Texture
    dim3 blk(32, 32);
    dim3 grid(divUp(m_image_width, blk.x), divUp(m_image_height, blk.y));
    device::RGBDToRGBKernel<<<grid, blk, 0, stream>>>(
        rgbd_tex, m_g_rgb.ptr(), m_image_width, m_image_height);

    // Copy from device to host
    cudaSafeCall(hipMemcpyAsync(
        m_h_rgb, m_g_rgb.ptr(),
        sizeof(uchar3) * m_image_width * m_image_height,
        hipMemcpyDeviceToHost, stream));
    cudaSafeCall(hipStreamSynchronize(stream));

    cv::Mat image(m_image_height, m_image_width, CV_8UC3, m_h_rgb);
    // Convert to gray
    cv::Mat gray_image;
    cv::cvtColor(image, gray_image, cv::COLOR_RGB2GRAY);
    // Detect ORB features
    std::vector<cv::KeyPoint> keypoints;
    cv::Ptr<cv::Feature2D> orb = cv::ORB::create();
    orb->detectAndCompute(gray_image, cv::Mat(), keypoints, descriptor);
    // Save keypoints into mat
    keypoint = cv::Mat(keypoints.size(), 1, CV_32FC2);
    for (int i = 0; i < keypoints.size(); i++)
    {
        keypoint.at<float2>(i, 0) = make_float2(keypoints[i].pt.x, keypoints[i].pt.y);
    }

    // Log info
    LOG(INFO) << "Number of keypoints detected: " << keypoints.size();
}