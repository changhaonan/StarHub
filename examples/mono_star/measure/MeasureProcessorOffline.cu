#include "hip/hip_runtime.h"
#include "MeasureProcessorOffline.h"
#include <star/img_proc/image_resize.cuh>

star::MeasureProcessorOffline::MeasureProcessorOffline()
{
	auto &config = ConfigParser::Instance();

	m_fetcher = std::make_shared<VolumeDeformFileFetch>(config.data_path());
	m_surfel_map = std::make_shared<SurfelMap>(config.downsample_img_cols(), config.downsample_img_rows());
	m_surfel_map_initializer = std::make_shared<SurfelMapInitializer>(
		config.raw_img_cols(),
		config.raw_img_rows(),
		config.clip_near(),
		config.clip_far(),
		config.surfel_radius_scale(),
		config.depth_intrinsic_raw());

	m_start_frame_idx = config.start_frame_idx();
	m_step_frame = config.step_frame();

	// Camera-related
	m_downsample_scale = config.downsample_scale();

	// Allocate buffer
	size_t num_pixel = config.raw_img_cols() * config.raw_img_rows();
	m_g_raw_color_img.create(num_pixel);
	m_g_raw_depth_img.create(num_pixel);

	cudaSafeCall(hipHostMalloc((void **)&m_raw_depth_img_buff, num_pixel * sizeof(unsigned short)));
	cudaSafeCall(hipHostMalloc((void **)&m_raw_color_img_buff, num_pixel * sizeof(uchar3)));
}

star::MeasureProcessorOffline::~MeasureProcessorOffline()
{
	m_g_raw_color_img.release();
	m_g_raw_depth_img.release();

	cudaSafeCall(hipHostFree(m_raw_depth_img_buff));
	cudaSafeCall(hipHostFree(m_raw_color_img_buff));
}

void star::MeasureProcessorOffline::Process(
	StarStageBuffer &star_stage_buffer_this,
	const StarStageBuffer &star_stage_buffer_prev,
	hipStream_t stream,
	const unsigned frame_idx)
{

	// Do nothing now
}

void star::MeasureProcessorOffline::processFrame(
	const unsigned frame_idx,
	hipStream_t stream)
{
	// 1. Load a rgb image && depth image
	const auto image_idx = size_t(frame_idx) * m_step_frame + m_start_frame_idx;
	m_fetcher->FetchRGBImage(0, image_idx, m_raw_color_img);
	m_fetcher->FetchDepthImage(0, image_idx, m_raw_depth_img);
	
	// CPU copy
	memcpy(m_raw_color_img_buff, m_raw_color_img.data,
        sizeof(uchar3) * m_raw_color_img.total()
    );
	memcpy(m_raw_depth_img_buff, m_raw_depth_img.data,
        sizeof(unsigned short) * m_raw_depth_img.total()
    );
	
	// Copy to GPU
	cudaSafeCall(hipMemcpyAsync(
		m_g_raw_color_img.ptr(),
		m_raw_color_img_buff,
		m_raw_color_img.total() * sizeof(uchar3),
		hipMemcpyHostToDevice,
		stream));
	cudaSafeCall(hipMemcpyAsync(
		m_g_raw_depth_img.ptr(),
		m_raw_depth_img_buff,
		m_raw_depth_img.total() * sizeof(unsigned short),
		hipMemcpyHostToDevice,
		stream));

	// 2. Initialize surfel map
	m_surfel_map_initializer->InitFromRGBDImage(
		GArrayView(m_g_raw_color_img),
		GArrayView(m_g_raw_depth_img),
		frame_idx,
		*m_surfel_map,
		stream);
	cudaSafeCall(hipStreamSynchronize(stream));

	// 3. Visualize
	saveContext(frame_idx, stream);
}

void star::MeasureProcessorOffline::saveContext(
	const unsigned frame_idx,
	hipStream_t stream)
{
	// Prepare
	auto &context = easy3d::Context::Instance();

	// Draw origin
	drawOrigin();

	// Draw point cloud
	context.addPointCloud("point_cloud");
	visualize::SavePointCloud(m_surfel_map->VertexConfidReadOnly(), context.at("point_cloud"));

	context.addPointCloud("color_cloud");
	visualize::SaveColoredPointCloud(
		m_surfel_map->VertexConfidReadOnly(),
		m_surfel_map->ColorTimeReadOnly(),
		context.at("color_cloud"));

	context.addPointCloud("normal_cloud", "", Eigen::Matrix4f::Identity(), 0.5f, "shadow");
	visualize::SavePointCloudWithNormal(
		m_surfel_map->VertexConfidReadOnly(),
		m_surfel_map->NormalRadiusReadOnly(),
		context.at("normal_cloud"));
}

void star::MeasureProcessorOffline::drawOrigin()
{
	auto &config = ConfigParser::Instance();
	auto &context = easy3d::Context::Instance();

	// Draw Tsdf area
	Eigen::Matrix4f bb_center = Eigen::Matrix4f::Identity();
	float3 origin = config.tsdf_origin();
	float voxel_size = config.tsdf_voxel_size();
	float box_width = voxel_size * float(config.tsdf_width());
	float box_height = voxel_size * float(config.tsdf_height());
	float box_depth = voxel_size * float(config.tsdf_depth());
	bb_center(0, 3) = origin.x + box_width / 2.f;
	bb_center(1, 3) = origin.y + box_height / 2.f;
	bb_center(2, 3) = origin.z + box_depth / 2.f;
	context.addBoundingBox("bounding_box", "helper", bb_center, box_width, box_height, box_depth);
	context.addCoord("origin", "helper", Eigen::Matrix4f::Identity(), 1.f);

	std::string cam_name = "cam_0";
	context.addCamera(cam_name, cam_name, config.extrinsic()[0]);
}