#include "hip/hip_runtime.h"
#include "MeasureProcessorOffline.h"

star::MeasureProcessorOffline::MeasureProcessorOffline()
{
	ConfigParser &config_parser = ConfigParser::Instance();
	std::cout << config_parser.data_prefix() << std::endl;
	m_fetcher = std::make_shared<VolumeDeformFileFetch>(config_parser.data_path());
	m_surfel_map = std::make_shared<SurfelMap>(config_parser.downsample_img_cols(), config_parser.downsample_img_rows());
	m_surfel_map_initializer = std::make_shared<SurfelMapInitializer>(
		config_parser.raw_img_cols(),
		config_parser.raw_img_rows(),
		config_parser.clip_near(),
		config_parser.clip_far(),
		config_parser.surfel_radius_scale(),
		config_parser.depth_intrinsic_raw());

	m_start_frame_idx = config_parser.start_frame_idx();
	m_step_frame = config_parser.step_frame();

	// Camera-related
	m_downsample_scale = config_parser.downsample_scale();

	// Allocate buffer
	size_t num_pixel = config_parser.raw_img_cols() * config_parser.raw_img_rows();
	m_g_raw_color_img.create(num_pixel);
	m_g_raw_depth_img.create(num_pixel);

	cudaSafeCall(hipHostMalloc((void **)&m_raw_depth_img_buff, num_pixel * sizeof(unsigned short)));
	cudaSafeCall(hipHostMalloc((void **)&m_raw_color_img_buff, num_pixel * sizeof(uchar3)));
}

star::MeasureProcessorOffline::~MeasureProcessorOffline()
{
	m_g_raw_color_img.release();
	m_g_raw_depth_img.release();

	cudaSafeCall(hipHostFree(m_raw_depth_img_buff));
	cudaSafeCall(hipHostFree(m_raw_color_img_buff));
}

void star::MeasureProcessorOffline::Process(
	StarStageBuffer &star_stage_buffer_this,
	const StarStageBuffer &star_stage_buffer_prev,
	hipStream_t stream,
	const unsigned frame_idx)
{

	// Do nothing now
}

void star::MeasureProcessorOffline::processFrame(
	const unsigned frame_idx,
	hipStream_t stream)
{
	// 1. Load a rgb image && depth image
	const auto image_idx = size_t(frame_idx) * m_step_frame + m_start_frame_idx;
	m_fetcher->FetchRGBImage(0, image_idx, m_raw_color_img);
	m_fetcher->FetchDepthImage(0, image_idx, m_raw_depth_img);
	
	// CPU copy
	memcpy(m_raw_color_img_buff, m_raw_color_img.data,
        sizeof(uchar3) * m_raw_color_img.total()
    );
	memcpy(m_raw_depth_img_buff, m_raw_depth_img.data,
        sizeof(unsigned short) * m_raw_depth_img.total()
    );
	
	// Copy to GPU
	cudaSafeCall(hipMemcpyAsync(
		m_g_raw_color_img.ptr(),
		m_raw_color_img_buff,
		m_raw_color_img.total() * sizeof(uchar3),
		hipMemcpyHostToDevice,
		stream));
	cudaSafeCall(hipMemcpyAsync(
		m_g_raw_depth_img.ptr(),
		m_raw_depth_img_buff,
		m_raw_depth_img.total() * sizeof(unsigned short),
		hipMemcpyHostToDevice,
		stream));

	// 2. Initialize surfel map
	m_surfel_map_initializer->InitFromRGBDImage(
		GArrayView(m_g_raw_color_img),
		GArrayView(m_g_raw_depth_img),
		frame_idx,
		*m_surfel_map,
		stream);
	cudaSafeCall(hipStreamSynchronize(stream));

	// 3. Visualize
	saveContext(frame_idx, stream);
}

void star::MeasureProcessorOffline::saveContext(
	const unsigned frame_idx,
	hipStream_t stream)
{
	// Prepare
	auto &context = easy3d::Context::Instance();

	// Draw origin
	drawOrigin();

	// Draw point cloud
	context.addPointCloud("point_cloud");
	visualize::SavePointCloud(m_surfel_map->VertexConfidReadOnly(), context.at("point_cloud"));

	context.addPointCloud("color_cloud");
	visualize::SaveColoredPointCloud(
		m_surfel_map->VertexConfidReadOnly(),
		m_surfel_map->ColorTimeReadOnly(),
		context.at("color_cloud"));

	context.addPointCloud("normal_cloud", "", Eigen::Matrix4f::Identity(), 0.5f, "shadow");
	visualize::SavePointCloudWithNormal(
		m_surfel_map->VertexConfidReadOnly(),
		m_surfel_map->NormalRadiusReadOnly(),
		context.at("normal_cloud"));
}

void star::MeasureProcessorOffline::drawOrigin()
{
	auto &config_parser = ConfigParser::Instance();
	auto &context = easy3d::Context::Instance();

	// Draw Tsdf area
	Eigen::Matrix4f bb_center = Eigen::Matrix4f::Identity();
	float3 origin = config_parser.tsdf_origin();
	float voxel_size = config_parser.tsdf_voxel_size();
	float box_width = voxel_size * float(config_parser.tsdf_width());
	float box_height = voxel_size * float(config_parser.tsdf_height());
	float box_depth = voxel_size * float(config_parser.tsdf_depth());
	bb_center(0, 3) = origin.x + box_width / 2.f;
	bb_center(1, 3) = origin.y + box_height / 2.f;
	bb_center(2, 3) = origin.z + box_depth / 2.f;
	context.addBoundingBox("bounding_box", "helper", bb_center, box_width, box_height, box_depth);
	context.addCoord("origin", "helper", Eigen::Matrix4f::Identity(), 1.f);

	std::string cam_name = "cam_0";
	context.addCamera(cam_name, cam_name, config_parser.extrinsic()[0]);
}