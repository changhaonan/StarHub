#include "hip/hip_runtime.h"
#include "star/common/global_configs.h"
#include "common/sanity_check.h"
#include "star/common/Macros.h"
#include "star/types/term_offset_types.h"
#include "star/warp_solver/utils/solver_encode.h"
#include "star/warp_solver/NodePair2TermsIndex.h"
#include <>

namespace star { namespace device {

	__host__ __device__ __forceinline__
		void computeKVNodePairKNN(
			const unsigned short* __restrict__ knn_arr,
			unsigned* __restrict__ nodepair_key
		) {
		auto offset = 0;
		for (auto i = 0; i < d_surfel_knn_size; i++) {
			const auto node_i = knn_arr[i];
			for (auto j = 0; j < d_surfel_knn_size; j++) {
				const auto node_j = knn_arr[j];
				if (node_i < node_j) {
					nodepair_key[offset] = encode_nodepair(node_i, node_j);
					offset++;
				}
			}
		}
	}


	__global__ void buildKeyValuePairKernel(
		const unsigned short* __restrict__ dense_image_knn_patch,
		const ushort3* __restrict__ node_graph,
		const unsigned node_size,
		const unsigned short* __restrict__ sparse_feature_knn_patch,
		const TermTypeOffset offset,
		unsigned* __restrict__ nodepair_keys,
		unsigned* __restrict__ term_values
	) {
		const auto term_idx = threadIdx.x + blockIdx.x * blockDim.x;
		TermType term_type;
		unsigned typed_term_idx, kv_offset;
		query_nodepair_index(term_idx, offset, term_type, typed_term_idx, kv_offset);

		// Compute the pair key locally
		unsigned term_nodepair_key[d_surfel_knn_pair_size];
		unsigned save_size = d_surfel_knn_pair_size;

		// Zero init
#pragma unroll
		for (auto i = 0; i < d_surfel_knn_pair_size; i++) {
			term_nodepair_key[i] = UNINITIALIZED_KEY;
		}

		switch (term_type) {
		case TermType::DenseImage:
			computeKVNodePairKNN(dense_image_knn_patch + d_surfel_knn_size * typed_term_idx, term_nodepair_key);
			break;
		case TermType::Reg:
			{
				const auto node_pair = node_graph[typed_term_idx];
				if (node_pair.x < node_pair.y) {
					term_nodepair_key[0] = encode_nodepair(node_pair.x, node_pair.y);
				}
				else if (node_pair.y < node_pair.x) {
					term_nodepair_key[0] = encode_nodepair(node_pair.y, node_pair.x);
				}
				save_size = 1;
			}
			break;
		case TermType::NodeTranslation:
			save_size = 0;  // No pairs here
			break;
		case TermType::Feature:
			computeKVNodePairKNN(sparse_feature_knn_patch + d_surfel_knn_size * typed_term_idx, term_nodepair_key);
			break;
		default:
			save_size = 0;
			break;
		}

		// Save it
		for (auto i = 0; i < save_size; i++) {
			nodepair_keys[kv_offset + i] = term_nodepair_key[i];
			term_values[kv_offset + i] = term_idx;
		}
	}


	__global__ void segmentNodePairKernel(
		const GArrayView<unsigned> sorted_node_pair,
		unsigned* segment_label
	) {
		// Check the valid of node size
		const auto idx = threadIdx.x + blockDim.x * blockIdx.x;
		if (idx >= sorted_node_pair.Size()) return;

		// The label must be written
		unsigned label = 0;

		// Check the size of node pair
		const auto encoded_pair = sorted_node_pair[idx];
		unsigned node_i, node_j;
		decode_nodepair(encoded_pair, node_i, node_j);
		if ((encoded_pair != UNINITIALIZED_KEY) && (node_i > d_max_num_nodes || node_j > d_max_num_nodes)) { //Take -1 into account
			// pass
		}
		else {
			if (idx == 0) label = 1;
			else // Can check the prev one
			{
				const auto encoded_prev = sorted_node_pair[idx - 1];
				if (encoded_prev != encoded_pair) label = 1;
			}
		}

		// Write to result
		segment_label[idx] = label;
	}


	__global__ void compactNodePairKeyKernel(
		const GArrayView<unsigned> sorted_node_pair,
		const unsigned* segment_label,
		const unsigned* inclusive_sum_label,
		unsigned* compacted_key,
		unsigned* compacted_offset,
		bool* negative_exist
	) {
		const auto idx = threadIdx.x + blockDim.x * blockIdx.x;
		if (idx < sorted_node_pair.Size() - 1) {
			if (segment_label[idx] > 0) {
				const auto compacted_idx = inclusive_sum_label[idx] - 1;
				compacted_key[compacted_idx] = sorted_node_pair[idx];
				compacted_offset[compacted_idx] = idx;
			}
		}
		else if (idx == sorted_node_pair.Size() - 1) {
			//The size of the sorted_key, segment label and 
			//inclusive-sumed segment are the same
			if (sorted_node_pair[idx] == UNINITIALIZED_KEY) {
				*negative_exist = true;
			}
			else {
				const auto last_idx = inclusive_sum_label[idx];
				compacted_offset[last_idx] = sorted_node_pair.Size();  // Will this be a problem?
				if (segment_label[idx] > 0) {
					const auto compacted_idx = last_idx - 1;
					compacted_key[compacted_idx] = sorted_node_pair[idx];
					compacted_offset[compacted_idx] = idx;
				}
				*negative_exist = false;
			}
		}
	}

	__global__ void computeSymmetricNodePairKernel(
		const GArrayView<unsigned> compacted_key,
		const unsigned* compacted_offset,
		unsigned* full_nodepair_key,
		uint2* full_term_start_end
	) {
		const auto idx = threadIdx.x + blockDim.x * blockIdx.x;
		if (idx < compacted_key.Size()) {
			const unsigned nodepair = compacted_key[idx];
			const unsigned start_idx = compacted_offset[idx];
			const unsigned end_idx = compacted_offset[idx + 1]; //This is safe
			unsigned node_i, node_j;
			decode_nodepair(nodepair, node_i, node_j);
			const unsigned sym_nodepair = encode_nodepair(node_j, node_i);
			//printf("i: %d, j: %d.\n", node_i, node_j);
			full_nodepair_key[2 * idx + 0] = nodepair;
			full_nodepair_key[2 * idx + 1] = sym_nodepair;
			full_term_start_end[2 * idx + 0] = make_uint2(start_idx, end_idx);
			full_term_start_end[2 * idx + 1] = make_uint2(start_idx, end_idx);
		}
	}

} // namespace device
} // namespace star

void star::NodePair2TermsIndex::buildTermKeyValue(hipStream_t stream) {
	// Correct the size of array
	const auto num_kvs = NumKeyValuePairs();
	m_nodepair_keys.ResizeArrayOrException(num_kvs);
	m_term_idx_values.ResizeArrayOrException(num_kvs);

	const auto num_terms = NumTerms();
	dim3 blk(256);
	dim3 grid(divUp(num_terms, blk.x));
	device::buildKeyValuePairKernel<<<grid, blk, 0, stream>>>(
		m_term2node.dense_image_knn_patch.Ptr(),
		m_term2node.node_graph.Ptr(),
		m_term2node.node_size,
		m_term2node.sparse_feature_knn_patch.Ptr(),
		m_term_offset,
		m_nodepair_keys.Ptr(),
		m_term_idx_values.Ptr()
	);

	// Sync and check error
#if defined(CUDA_DEBUG_SYNC_CHECK)
	cudaSafeCall(hipStreamSynchronize(stream));
	cudaSafeCall(hipGetLastError());
#endif
}

void star::NodePair2TermsIndex::sortCompactTermIndex(hipStream_t stream) {
	m_nodepair2term_sorter.Sort(m_nodepair_keys.View(), m_term_idx_values.View(), 24, stream);

	// Do segmentation
	m_segment_label.ResizeArrayOrException(m_nodepair_keys.ArraySize());
	GArrayView<unsigned> sorted_node_pair(m_nodepair2term_sorter.valid_sorted_key);
	dim3 blk(256);
	dim3 grid(divUp(sorted_node_pair.Size(), blk.x));
	device::segmentNodePairKernel<<<grid, blk, 0, stream>>>(sorted_node_pair, m_segment_label.Ptr());

	// Do prefix sum and compaction
	m_segment_label_prefixsum.InclusiveSum(m_segment_label.View(), stream);

	// Create -1 flag
	bool* negative_exist;
	cudaSafeCall(hipMalloc((void**)&negative_exist, sizeof(bool)));
	device::compactNodePairKeyKernel<<<grid, blk, 0, stream>>>(
		sorted_node_pair,
		m_segment_label.Ptr(),
		m_segment_label_prefixsum.valid_prefixsum_array.ptr(),
		m_half_nodepair_keys.Ptr(),
		m_half_nodepair2term_offset.Ptr(),
		negative_exist
	);
	bool exist_flag;
	cudaSafeCall(hipMemcpyAsync(&exist_flag, negative_exist, sizeof(bool), hipMemcpyDeviceToHost, stream));

	// Sync and check error
#if defined(CUDA_DEBUG_SYNC_CHECK)
	cudaSafeCall(hipStreamSynchronize(stream));
	cudaSafeCall(hipGetLastError());
#endif

	QueryValidNodePairSize(exist_flag, stream); //Resize, blocking
	
	// Debug
	//CompactedIndexLog();
}

void star::NodePair2TermsIndex::buildSymmetricCompactedIndex(hipStream_t stream) {
	//Assume the size has been queried
	dim3 blk(128);
	dim3 grid(divUp(m_half_nodepair_keys.ArraySize(), blk.x));
	device::computeSymmetricNodePairKernel<<<grid, blk, 0, stream>>>(
		m_half_nodepair_keys.View(),
		m_half_nodepair2term_offset.Ptr(),
		m_compacted_nodepair_keys.Ptr(),
		m_nodepair_term_range.Ptr()
	);

	//Sort the key-value pair
	m_symmetric_kv_sorter.Sort(m_compacted_nodepair_keys.View(), m_nodepair_term_range.View(), 24, stream);

	//Sync and check error
#if defined(CUDA_DEBUG_SYNC_CHECK)
	cudaSafeCall(hipStreamSynchronize(stream));
	cudaSafeCall(hipGetLastError());
#endif
}

void star::NodePair2TermsIndex::QueryValidNodePairSize(const bool negative_exist, hipStream_t stream) {
	const unsigned* num_unique_pair_dev = m_segment_label_prefixsum.valid_prefixsum_array.ptr() + (m_segment_label_prefixsum.valid_prefixsum_array.size() - 1);
	unsigned num_unique_pair;
	cudaSafeCall(hipMemcpyAsync(&num_unique_pair, num_unique_pair_dev, sizeof(unsigned), hipMemcpyDeviceToHost, stream));
	cudaSafeCall(hipStreamSynchronize(stream));

	if (negative_exist) num_unique_pair -= 1;  //Remove -1 part
	//Correct the size
	m_half_nodepair_keys.ResizeArrayOrException(num_unique_pair);
	m_half_nodepair2term_offset.ResizeArrayOrException(num_unique_pair + 1);
	m_compacted_nodepair_keys.ResizeArrayOrException(2 * num_unique_pair);
	m_nodepair_term_range.ResizeArrayOrException(2 * num_unique_pair);
}

void star::NodePair2TermsIndex::CompactedIndexLog() {
	// Check
	std::vector<unsigned> h_nodepair_keys;
	h_nodepair_keys.resize(m_nodepair2term_sorter.valid_sorted_key.size());
	std::vector<unsigned> h_term_idx_values;
	h_term_idx_values.resize(m_nodepair2term_sorter.valid_sorted_value.size());
	m_nodepair2term_sorter.valid_sorted_key.download(h_nodepair_keys.data());
	m_nodepair2term_sorter.valid_sorted_value.download(h_term_idx_values.data());

	std::vector<unsigned> h_half_nodepair_keys;
	std::vector<unsigned> h_half_nodepair2term_offset;
	m_half_nodepair_keys.View().Download(h_half_nodepair_keys);
	m_half_nodepair2term_offset.View().Download(h_half_nodepair2term_offset);

	 printf("-------------------Check----------------\n");
	 for(auto i = 0; i < h_half_nodepair_keys.size(); ++i) {
		 for(auto j = h_half_nodepair2term_offset[i]; j < h_half_nodepair2term_offset[i+1]; ++j) {
			 printf("term: %d, code: %d, offset: %d - %d\n",
				 h_term_idx_values[j], h_half_nodepair_keys[i],
				 h_half_nodepair2term_offset[i],
				 h_half_nodepair2term_offset[i+1]);
		 }
	 }

	 printf("-------------------Raw-------------------\n");
	 for(auto i = 0; i < h_term_idx_values.size(); ++i) {
		 printf("offset: %d | code: %d, term: %d\n",
			 i, h_nodepair_keys[i], h_term_idx_values[i]
		 );
	 }
	 printf("-------------------Finish---------------\n");

	//Debug
	std::vector<unsigned> sorted_key;
	m_nodepair_keys.View().Download(sorted_key);
	std::cout << sorted_key[sorted_key.size() - 1] << std::endl;
}