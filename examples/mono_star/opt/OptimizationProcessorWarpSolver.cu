#include <mono_star/common/ConfigParser.h>
#include <mono_star/opt/OptimizationProcessorWarpSolver.h>

star::OptimizationProcessorWarpSolver::OptimizationProcessorWarpSolver()
{
	std::cout << "Initialize OptimizationProcessorWarpSolver..." << std::endl;
	auto &config = ConfigParser::Instance();
	m_num_cam = config.num_cam();

	for (auto cam_idx = 0; cam_idx < m_num_cam; ++cam_idx)
	{
		m_cam2world[cam_idx] = config.extrinsic()[cam_idx];
		m_image_height[cam_idx] = config.downsample_img_rows(cam_idx);
		m_image_width[cam_idx] = config.downsample_img_cols(cam_idx);
		m_intrinsic[cam_idx] = config.rgb_intrinsic_downsample(cam_idx);
	}

	m_warp_solver = std::make_shared<WarpSolver>(m_num_cam, m_image_height, m_image_width, m_intrinsic);
}

star::OptimizationProcessorWarpSolver::~OptimizationProcessorWarpSolver()
{
	m_warp_solver->ReleaseBuffer();
}

void star::OptimizationProcessorWarpSolver::ProcessFrame(
	Measure4Solver &measure4solver,
	Render4Solver &render4solver,
	Geometry4Solver &geometry4solver,
	NodeGraph4Solver &node_graph4solver,
	NodeFlow4Solver &nodeflow4solver,
	OpticalFlow4Solver &opticalflow4solver,
	KeyPoint4Solver &keypoint4solver,
	const unsigned frame_idx,
	hipStream_t stream)
{
	// Collect input
	printf("Optimizer Processing.\n");
	if (frame_idx == 0)
		return; // Operation start from frame 1
	// Set input
	m_warp_solver->SetSolverInputs(
		measure4solver,
		render4solver,
		geometry4solver,
		node_graph4solver,
		nodeflow4solver,
		opticalflow4solver,
		keypoint4solver,
		m_cam2world);

	// Seperate test
	bool opt_success = m_warp_solver->SolveStreamed();
	if (!opt_success)
		std::cout << "Optimization failed. Use previous result instead." << std::endl;

	cudaSafeCall(hipStreamSynchronize(stream));
}