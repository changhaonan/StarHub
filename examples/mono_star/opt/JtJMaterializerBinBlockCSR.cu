#include "hip/hip_runtime.h"
#include "pcg_solver/solver_configs.h"
#include "star/warp_solver/JtJMaterializer.h"
#include <>

namespace star { namespace device {

	__global__ void assembleBinBlockCSRKernel(
		const unsigned matrix_size,
		const float* diagonal_blks,
		const float* nondiagonal_blks,
		const int* csr_rowptr,
		const unsigned* blkrow_offset,
		float* JtJ_data
	) {
		const auto row_idx = threadIdx.x + blockDim.x * blockIdx.x;
		if (row_idx >= matrix_size) return;

		// Now the query should all be safe
		int data_offset = csr_rowptr[row_idx];
		const auto blkrow_idx = row_idx / d_node_variable_dim;
		const auto inblk_offset = row_idx % d_node_variable_dim;

		// First fill the diagonal blks
		for (auto k = 0; k < d_node_variable_dim; k++, data_offset += bin_size) {
			JtJ_data[data_offset] = diagonal_blks[d_node_variable_dim_square * blkrow_idx + inblk_offset + d_node_variable_dim * k];
		}

		// Next fill the non-diagonal blks
		auto Iij_begin = blkrow_offset[blkrow_idx];
		const auto Iij_end = blkrow_offset[blkrow_idx + 1];
		for (; Iij_begin < Iij_end; Iij_begin++) {
			for (int k = 0; k < d_node_variable_dim; k++, data_offset += bin_size) {
				JtJ_data[data_offset] = nondiagonal_blks[d_node_variable_dim_square * Iij_begin + inblk_offset + d_node_variable_dim * k];
			}
		}
	}

} // device
} // star

void star::JtJMaterializer::AssembleBinBlockCSR(star::GArrayView<float> diagonal_blks, hipStream_t stream) {
	// Zero out the matrix
	cudaSafeCall(hipMemsetAsync(m_binblock_csr_data.Ptr(), 0, sizeof(float) * m_binblock_csr_data.BufferSize(), stream));

	// The size of the matrix
	STAR_CHECK(diagonal_blks.Size() % d_node_variable_dim == 0);
	const auto matrix_size = diagonal_blks.Size() / d_node_variable_dim;

	dim3 assemble_blk(128);
	dim3 assemble_grid(divUp(matrix_size, assemble_blk.x));
	device::assembleBinBlockCSRKernel<<<assemble_grid, assemble_blk, 0, stream>>>(
		matrix_size,
		diagonal_blks.Ptr(),
		m_nondiag_blks.Ptr(),
		m_nodepair2term_map.binblock_csr_rowptr.Ptr(),
		m_nodepair2term_map.blkrow_offset.Ptr(),
		m_binblock_csr_data.Ptr()
	);

	// Sync and check error
#if defined(CUDA_DEBUG_SYNC_CHECK)
	cudaSafeCall(hipStreamSynchronize(stream));
	cudaSafeCall(hipGetLastError());
#endif

	// Construct the spmv applier
	m_spmv_handler->SetInputs(
		m_binblock_csr_data.Ptr(),
		m_nodepair2term_map.binblock_csr_rowptr.Ptr(),
		m_nodepair2term_map.binblock_csr_colptr,
		matrix_size
	);
}