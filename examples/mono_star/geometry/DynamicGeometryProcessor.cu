#include "hip/hip_runtime.h"
#include <star/geometry/node_graph/Skinner.h>
#include <star/geometry/node_graph/NodeGraphManipulator.h>
#include <star/geometry/surfel/SurfelNodeDeformer.h>
#include <star/visualization/Visualizer.h>
#include <mono_star/common/ConfigParser.h>
#include "DynamicGeometryProcessor.h"

namespace star::device
{
    __global__ void GenerateSegmentationMapKernel(
        const ucharX<d_max_num_semantic> *__restrict__ semantic_prob,
        hipTextureObject_t index_map,
        hipSurfaceObject_t segmentation_map,
        const unsigned num_valid_surfel,
        const unsigned img_rows,
        const unsigned img_cols)
    {
        const auto idx = threadIdx.x + blockIdx.x * blockDim.x;
        const auto idy = threadIdx.y + blockIdx.y * blockDim.y;
        if (idx >= img_cols || idy >= img_rows)
            return;

        const auto surfel_idx = tex2D<int>(index_map, idx, idy);
        if (surfel_idx == 0xFFFFFFFF || surfel_idx >= num_valid_surfel)
            return;
        auto surfel_semantic = semantic_prob[surfel_idx];
        surf2Dwrite(max_id(surfel_semantic), segmentation_map, idx * sizeof(int), idy);
    }

}

star::DynamicGeometryProcessor::DynamicGeometryProcessor()
{
    std::cout << "Initilize DynamicGeometryProcessor..." << std::endl;
    auto &config = ConfigParser::Instance();

    m_data_geometry = std::make_shared<star::SurfelGeometry>();
    m_model_geometry[0] = std::make_shared<star::SurfelGeometry>();
    m_model_geometry[1] = std::make_shared<star::SurfelGeometry>();
    m_node_graph[0] = std::make_shared<star::NodeGraph>(config.node_radius());
    m_node_graph[1] = std::make_shared<star::NodeGraph>(config.node_radius());

    // Render
    m_renderer = std::make_shared<Renderer>(
        config.num_cam(),
        config.downsample_img_cols(),
        config.downsample_img_rows(),
        config.rgb_intrinsic_downsample(),
        config.max_rendering_depth());

    m_renderer->MapDataSurfelGeometryToCuda(0, *m_data_geometry);
    m_renderer->MapModelSurfelGeometryToCuda(0, *m_model_geometry[0]);
    m_renderer->MapModelSurfelGeometryToCuda(1, *m_model_geometry[1]);

    // Camera-related
    m_num_cam = config.num_cam();
    m_cam2world = config.extrinsic()[0];
    m_intrinsic = config.rgb_intrinsic_downsample();
    m_img_cols = config.downsample_img_cols();
    m_img_rows = config.downsample_img_rows();
    // Regulation
    m_dynamic_regulation = config.dynamic_regulation();
    // Other
    m_enable_semantic_surfel = config.enable_semantic_surfel();
    m_reinit_counter = config.reinit_counter();

    m_geometry_fusor = std::make_shared<GeometryFusor>(
        m_model_geometry,
        m_node_graph,
        m_renderer,
        m_num_cam,
        m_img_cols,
        m_img_rows,
        m_cam2world,
        m_intrinsic,
        m_enable_semantic_surfel,
        m_reinit_counter,
        m_dynamic_regulation);
    // Map
    createInt32TextureSurface(m_img_rows, m_img_cols, m_segmentation_map);

    // Vis
    m_enable_vis = config.enable_vis();
    m_pcd_size = config.pcd_size();
    m_node_graph_size = config.graph_node_size();

    // Eval
    m_track_semantic_label = config.track_semantic_label();
}

star::DynamicGeometryProcessor::~DynamicGeometryProcessor()
{
    m_renderer->UnmapDataSurfelGeometryFromCuda(0);
    m_renderer->UnmapModelSurfelGeometryFromCuda(0);
    m_renderer->UnmapModelSurfelGeometryFromCuda(1);

    if (m_solver_maps_mapped)
        m_renderer->UnmapSolverMapsFromCuda();
    if (m_observation_maps_mapped)
        m_renderer->UnmapObservationMapsFromCuda();

    releaseTextureCollect(m_segmentation_map);
}

void star::DynamicGeometryProcessor::ProcessFrame(
    const SurfelMapTex &surfel_map,
    const GArrayView<DualQuaternion> &solved_se3,
    const unsigned frame_idx,
    hipStream_t stream)
{
    // Assign
    m_solved_se3 = solved_se3;

    // Update geometry
    if (frame_idx == 0)
    {
        initGeometry(surfel_map, m_cam2world, frame_idx, stream);
    }
    else if (frame_idx > 0)
    {
        updateGeometry(surfel_map, solved_se3, frame_idx, stream); // Apply warp
    }

    // Generate map from geometry
    drawRenderMaps(frame_idx, stream);
    computeSurfelMapTex();

    // Vis
    if (m_enable_vis)
        saveContext(frame_idx, stream);
}

void star::DynamicGeometryProcessor::initGeometry(
    const SurfelMapTex &surfel_map, const Eigen::Matrix4f &cam2world, const unsigned frame_idx, hipStream_t stream)
{
    // Init Surfel geometry
    SurfelGeometryInitializer::InitFromGeometryMap(
        *m_model_geometry[m_buffer_idx],
        surfel_map,
        cam2world,
        m_enable_semantic_surfel,
        stream);

    // Init NodeGraph
    m_node_graph[m_buffer_idx]->InitializeNodeGraphFromVertex(
        m_model_geometry[m_buffer_idx]->LiveVertexConfidenceReadOnly(), frame_idx, false, stream);
    m_node_graph[m_buffer_idx]->ResetNodeGraphConnection(stream);

    // Perform Skinning without semantic
    auto geometry4skinner = m_model_geometry[m_buffer_idx]->GenerateGeometry4Skinner();
    auto node_graph4skinner = m_node_graph[m_buffer_idx]->GenerateNodeGraph4Skinner();
    Skinner::PerformSkinningFromLive(geometry4skinner, node_graph4skinner, stream);

    // Update with skinning with semantic
    if (m_enable_semantic_surfel)
    {
        // Update semantic prob
        NodeGraphManipulator::UpdateNodeSemanticProb(
            m_model_geometry[m_buffer_idx]->SurfelKNN().View(),
            m_model_geometry[m_buffer_idx]->SemanticProbReadOnly(),
            m_node_graph[m_buffer_idx]->GetNodeSemanticProb(),
            m_node_graph[m_buffer_idx]->GetNodeSemanticProbVoteBuffer(),
            stream);
        // Update node connection
        m_node_graph[m_buffer_idx]->ComputeNodeGraphConnectionFromSemantic(m_dynamic_regulation, stream);
        // Update surfel connection
        Skinner::UpdateSkinnningConnection(geometry4skinner, node_graph4skinner, stream);
    }
}

void star::DynamicGeometryProcessor::updateGeometry(
    const SurfelMapTex &surfel_map,
    const GArrayView<DualQuaternion> &solved_se3,
    const unsigned frame_idx,
    hipStream_t stream)
{
    if (solved_se3.Size() == 0)
        return;

    // Apply the deformation
    SurfelNodeDeformer::ForwardWarpSurfelsAndNodes(
        m_node_graph[m_buffer_idx]->DeformAccess(), *m_model_geometry[m_buffer_idx], solved_se3, stream);

#ifdef ENABLE_POSE_EVAL
    // Update eval-related data
    m_node_graph[m_buffer_idx]->UpdateNodeDeformAcc(solved_se3, stream);
#endif

    // Init data geometry
    SurfelGeometryInitializer::InitFromGeometryMap(
        *m_data_geometry,
        surfel_map,
        m_cam2world,
        m_enable_semantic_surfel,
        stream);

    // Apply the geometry fusion
    m_geometry_fusor->Fuse(
        m_buffer_idx,
        frame_idx,
        surfel_map,
        m_data_geometry,
        stream);

    // Reanchor the geometry
    auto next_buffer_idx = (m_buffer_idx + 1) & 1;
    SurfelGeometry::ReAnchor(
        m_model_geometry[m_buffer_idx],
        m_model_geometry[next_buffer_idx],
        stream);
    NodeGraph::ReAnchor(
        m_node_graph[m_buffer_idx],
        m_node_graph[next_buffer_idx],
        stream);
    m_buffer_idx = next_buffer_idx;
}

void star::DynamicGeometryProcessor::computeSurfelMapTex()
{
    // Get solver-map
    m_surfel_map_tex.vertex_confid = m_solver_maps.vertex_confid_map[0];
    m_surfel_map_tex.normal_radius = m_solver_maps.normal_radius_map[0];
    m_surfel_map_tex.color_time = 0;
    // Get observation-map
    m_surfel_map_tex.rgbd = m_observation_maps.rgbd_map[0];
    m_surfel_map_tex.depth = 0;
    m_surfel_map_tex.index = m_observation_maps.index_map[0];
    m_surfel_map_tex.segmentation = m_segmentation_map.texture;
    // Num
    m_surfel_map_tex.num_valid_surfel = m_model_geometry[m_buffer_idx]->NumValidSurfels();
}

void star::DynamicGeometryProcessor::saveContext(const unsigned frame_idx, hipStream_t stream)
{
    auto &context = easy3d::Context::Instance();

    // Save Geometry
    unsigned last_buffer_idx = (m_buffer_idx + 1) % 2;
    unsigned vis_buffer_idx = (m_model_geometry[last_buffer_idx]->NumValidSurfels() > 0) ? last_buffer_idx : m_buffer_idx;
    // Exist valid last geo
    std::string ref_color_name = "ref_color";
    context.addPointCloud(ref_color_name, ref_color_name, m_cam2world.inverse(), m_pcd_size);
    visualize::SaveColoredPointCloud(
        m_model_geometry[vis_buffer_idx]->ReferenceVertexConfidenceReadOnly(),
        m_model_geometry[vis_buffer_idx]->ColorTimeReadOnly(),
        context.at(ref_color_name));

    std::string live_color_name = "live_color";
    context.addPointCloud(live_color_name, live_color_name, m_cam2world.inverse(), m_pcd_size);
    visualize::SaveColoredPointCloud(
        m_model_geometry[vis_buffer_idx]->LiveVertexConfidenceReadOnly(),
        m_model_geometry[vis_buffer_idx]->ColorTimeReadOnly(),
        context.at(live_color_name));

    // Save node graph
    context.addGraph("ref_graph", "", m_cam2world.inverse(), m_node_graph_size);
    visualize::SaveGraph(
        m_node_graph[vis_buffer_idx]->ReferenceNodeCoordinateReadOnly(),
        m_node_graph[vis_buffer_idx]->NodeKnnReadOnly(),
        context.at("ref_graph"));

    context.addGraph("live_graph", "", m_cam2world.inverse(), m_node_graph_size);
    visualize::SaveGraph(
        m_node_graph[vis_buffer_idx]->LiveNodeCoordinateReadOnly(),
        m_node_graph[vis_buffer_idx]->NodeKnnReadOnly(),
        context.at("live_graph"));

// #ifndef ENABLE_EFFICIENCY_MODE
    // Save semantic
    if (m_enable_semantic_surfel)
    {
        std::string semantic_pcd_name = "semantic_pcd";
        context.addPointCloud(semantic_pcd_name, semantic_pcd_name, m_cam2world.inverse(), m_pcd_size);
        visualize::SaveSemanticPointCloud(
            m_model_geometry[vis_buffer_idx]->LiveVertexConfidenceReadOnly(),
            m_model_geometry[vis_buffer_idx]->SemanticProbReadOnly(),
            visualize::default_semantic_color_dict,
            context.at(semantic_pcd_name));

        // Visualize for node graph
        std::string segmentation_graph_name = "segmentation_graph";
        context.addGraph(segmentation_graph_name, segmentation_graph_name, m_cam2world.inverse(), m_node_graph_size, 0.0, 1.0, true);

        // Transfer to color first
        std::vector<uchar3> node_vertex_color;
        visualize::Semantic2Color(
            m_node_graph[vis_buffer_idx]->GetNodeSemanticProbReadOnly(),
            visualize::default_semantic_color_dict,
            node_vertex_color);
        std::vector<float4> h_node_vertex;
        m_node_graph[vis_buffer_idx]->LiveNodeCoordinateReadOnly().Download(h_node_vertex);
        std::vector<ushortX<d_node_knn_size>> h_edges;
        m_node_graph[vis_buffer_idx]->NodeKnnReadOnly().Download(h_edges);
        std::vector<floatX<d_node_knn_size>> h_node_connect;
        m_node_graph[vis_buffer_idx]->NodeKnnConnectWeightReadOnly().Download(h_node_connect);
        visualize::SaveGraph(h_node_vertex, node_vertex_color, h_edges, h_node_connect, context.at(segmentation_graph_name));
    }
// #endif

    if (frame_idx > 0 && m_track_semantic_label > 0)
    {
        // Evaluate average dq when there is only one moving object
        // Select the first 6 nodes from semantic 3.
        unsigned num_node_selected = 6;
        DualQuaternion average_dq;
        float3 average_pos;
        computeAverageNodeDeform(vis_buffer_idx, m_track_semantic_label, num_node_selected, average_dq, average_pos, stream);
        auto average_mat = average_dq.se3_matrix();
        average_mat.trans = average_pos;
        context.addCoord("average_dq", "", m_cam2world.inverse() * Eigen::Matrix4f(average_mat), 0.3);
    }

    // Save images
    context.addImage("ref-rgb");
    context.addImage("ref-depth");
    visualize::SaveNormalizeRGBDImage(m_observation_maps.rgbd_map[0], context.at("ref-rgb"), context.at("ref-depth"));
}

void star::DynamicGeometryProcessor::drawRenderMaps(
    const unsigned frame_idx,
    hipStream_t stream)
{
    // Generate new solver map (reference map inside)
    drawSolverMaps(
        frame_idx,
        m_buffer_idx,
        stream);
    // Generate new observation map (reference map inside)
    drawObservationMaps(
        frame_idx,
        m_buffer_idx,
        stream);
    // Generate new segmentation map
    drawSegmentationMap(
        frame_idx,
        m_buffer_idx,
        stream);
}

void star::DynamicGeometryProcessor::computeAverageNodeDeform(
    const unsigned buffer_idx,
    const unsigned short semantic_selected,
    const unsigned num_node_selected,
    DualQuaternion &average_node_deform,
    float3 &average_node_pos,
    hipStream_t stream)
{
    // Fetch node list
    std::vector<unsigned short> node_list_selected;
    NodeGraphManipulator::SelectNodeBySemantic(
        m_node_graph[buffer_idx]->GetNodeSemanticProbReadOnly(),
        semantic_selected,
        num_node_selected,
        node_list_selected);

    // Compute average dq
    NodeGraphManipulator::AvergeNodeMovementAndPos(
        m_node_graph[buffer_idx]->LiveNodeCoordinateReadOnly(),
        m_solved_se3,
        node_list_selected,
        m_node_graph[buffer_idx]->NodeDeformAcc(),
        average_node_deform,
        average_node_pos);
}

void star::DynamicGeometryProcessor::drawSolverMaps(
    const unsigned frame_idx,
    const unsigned geometry_idx,
    hipStream_t stream)
{
    // Generate new reference map
    m_renderer->UnmapModelSurfelGeometryFromCuda(geometry_idx, stream);
    cudaSafeCall(hipStreamSynchronize(stream));
    m_renderer->DrawSolverMapsWithRecentObservation(
        m_model_geometry[geometry_idx]->NumValidSurfels(),
        geometry_idx,
        0,
        frame_idx,
        m_cam2world.inverse());

    if (!m_solver_maps_mapped)
    {
        m_renderer->MapSolverMapsToCuda(m_solver_maps, stream);
        m_solver_maps_mapped = true;
    }
    m_renderer->MapModelSurfelGeometryToCuda(geometry_idx, stream);
    cudaSafeCall(hipStreamSynchronize(stream));
}

void star::DynamicGeometryProcessor::drawObservationMaps(
    const unsigned frame_idx,
    const unsigned geometry_idx,
    hipStream_t stream)
{
    // Generate new reference map
    m_renderer->UnmapModelSurfelGeometryFromCuda(geometry_idx, stream);
    cudaSafeCall(hipStreamSynchronize(stream));
    m_renderer->DrawObservationMaps(
        m_model_geometry[geometry_idx]->NumValidSurfels(),
        geometry_idx, 0, frame_idx,
        m_cam2world.inverse(),
        true);

    if (!m_observation_maps_mapped)
    {
        m_renderer->MapObservationMapsToCuda(m_observation_maps, stream);
        m_observation_maps_mapped = true;
    }
    m_renderer->MapModelSurfelGeometryToCuda(geometry_idx, stream);
    cudaSafeCall(hipStreamSynchronize(stream));
}

void star::DynamicGeometryProcessor::drawSegmentationMap(
    const unsigned frame_idx, const unsigned geometry_idx, hipStream_t stream)
{
    dim3 blk(32, 32);
    dim3 grid(divUp(m_img_rows, blk.x), divUp(m_img_cols, blk.y));
    device::GenerateSegmentationMapKernel<<<grid, blk, 0, stream>>>(
        m_model_geometry[geometry_idx]->SemanticProbReadOnly().Ptr(),
        m_observation_maps.index_map[0],
        m_segmentation_map.surface,
        m_model_geometry[geometry_idx]->NumValidSurfels(),
        m_img_rows,
        m_img_rows);
    cudaSafeCall(hipStreamSynchronize(stream));
}